#include "hip/hip_runtime.h"
#define MEMBRANE	//膜蒸发 //#define CONTACTANGLE	//接触角模型 _设置相应的流场
#define PTCALCFIVEPOINT //五点差分计算化学势 //#define PTCALCSEVENPOINT //七点差分计算化学势
//#define SIMPLECHEMBOUNDARY	 //简单化学势边界
#define COMPLEXCHEMBOUNDARY	 //带梯度的化学势边界

//#define DROPLETSPLASH

typedef int err_type;

#ifdef WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#include <cassert>
#include <string>
#include <cstdlib>
#include <sstream> 
#include <fstream> 
#include <iomanip>
#include <iostream> 
#include <algorithm>
#include <chrono>
#include <ctime>

#ifdef WIN32
#include <io.h>  //win
#include <direct.h>
#else
#include <unistd.h>
#include <sys/stat.h>
#endif


#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h> 
#include "hip/hip_runtime_api.h"
#include "ordered_tec.h"
#include "Common.h"
#include "utility.h"

using namespace std;
using namespace ORDERED_TEC;

void MembraneParaInit();
void MembraneParaShow();
void ContactAngleShow();
void MembraneParaFree();
void MembraneERCalc();
void AddFilm();

__global__ void SetFilm(char * Type, double * Dens, double * Pote);  
__global__ void SetPlate(char * Type, double * Dens, double * Pote); //更改底板润湿性
__global__ void SetPorePote(char *Type, double *Pote);
__global__ void SetHotLiquid(char * Type, double * Dens);

err_type LoadCheckPoint(char * FileName);
err_type SaveCheckPoint(const string TestName);

//作图函数
void plotInitField_origin();
void plotEosPressure(const string TestName);
void plotER_Time();
void plotER_PoreR();

__global__ void TemperatureGradient(char *Type, double *Te, double *Tx, double *Ty, double *Tz, double *Td, double *MVx, double *MVy, double *MVz, double *DVx, double *DVy, double *DVz);
__global__ void Temperature(char *Type, double *Dens, double *Te, double *MVx, double *MVy, double *MVz ,double *Tx , double *Ty , double *Tz, double *Td , double *DVx, double *DVy, double *DVz ) ;
//******************************************SchemeNumber*******************************************
// const int DROPLETSPLASH_FIELD_INIT = 0;
// const int CONTACTANGLE_FIELD_INIT = 1;


//******************************************ErrorCheck*******************************************
int err_den = false, err_distribution = false; 
//******************************************CheckPoint*******************************************
bool LoadInitFlag = false, SaveCheckPointFlag = false;
//*************************************************************************************************
struct DVector
{
	double x, y, z;
};



short(*Mxyz)[DXYZ] = NULL;
double *HostMelo = NULL, *HostDens = NULL, *HostPote = NULL, *HostTe = NULL; 
double Ts[DQ], Mass, ReTau, Tau, Viscosity, Radius, MaxSpeed, Width;
double Diameter, Gravity;
double DimLen, DimTime, DimMass, DimF, DimT, DimV, DimA, DimW, DimB;
double GravityX = 0, GravityY = 0, GravityZ = 0;
double K1, K2;
double SigmaI, SigmaL, deltaP, s, Laplace = 0;
double DenG, DenL, A, B, K, Ka, T, Tr, Tc, Rc, Pc, BasePt, BasePt2, CaLeft, CaCap=0, CaRight, DropVy, XLeft, XLeft_min= DX /2, XRight, XRight_max= DX / 2, DropMass=0,Mx= 0, My= 0, Mz= 0,My2,Mz2, XLeft0, XRight0, Hdown,starttime,endtime;//Hdown为液滴最低点高度
int    TasKNum = 1, No, NowStep, DropStep, AllStep, ShowStep, SaveStep, BeginTime, StepTime, LastTime, MModel, FModel;
int	   SpreadTime=0;

//******************************************MembranePara*******************************************
int PoreBottom, PoreTop, PoreCenterX, PoreCenterY, PoreRadius, FilmThickness;
int HotLiquidBottom, HotLiquidTop, ColdLiquidBottom, ColdLiquidTop, HotLiquidThickness, ColdLiquidThickness;
int PtRadius;
double HydroPhobicPt, HydroPhilicPt;
double Tcold, Thot;

__constant__ int _PoreBottom, _PoreTop, _PoreCenterX, _PoreCenterY, _PoreRadius, _PtRadius;
__constant__ int _HotLiquidBottom, _HotLiquidTop, _ColdLiquidBottom, _ColdLiquidTop;
__constant__ double _HydroPhobicPt,  _HydroPhilicPt; //疏水 ， 亲水
__constant__ double _Thot, _Tcold, _Tc;

//蒸发率计算相关
double ER,ER_LMH,ER_inst;	
double DimEvaporation;   
int startStep;
double startUpperMass = 0, NowUpperMass, preStepUpperMass = 0;
double Porosity; //孔隙率 
//追踪质量变化
double TotaldeltaMass , deltaMass;


//******************************************MembraneDataStructure*******************************************
double *Te, *Tx, *Ty, *Tz, *Td;
double *DVx, *DVy, *DVz;
//*************************************************************************************************

hipError_t  err;
char    *Type;
double  *Dens, *Pote, *Dist, *Temp, *Vx, *Vy, *Vz, *MVx, *MVy, *MVz, *Fx, *Fy, *Fz;
double(*TDen)[DZ];
__constant__ double _K, _A, _B, _T, _Ka, _Kq, _DenG, _DenL, _Tau, _Ts[DQ], _ReTau, _NowStep, _DropStep, _DimA, _Gravity, _Radius, _Width,  _BasePt, _BasePt2, _K1, _K2;//_interfaceDen[DXY]

const int DR = 5;
const int LX = DX / DR, LY = DY / DR, LZ = DZ / DR;
const int LXYZ = LX*LY*LZ;

void Initialize();
void SetMultiphase();
void ShowData();
void CalcMacroCPU();
//void DataShowSave();

void SaveVel();//保存底板K=2液滴底面的速度
void SaveY_Vel();//流场中每一个X-Z平面的Y速度 
void SavePointVel();//保存每一时刻边缘亲水点的最大速度
void SaveAllByType();
void SaveSpreadtime();
void SaveDataOriDen();
void SaveContactAngle();
void Save_interfaceDen();//测试表面图案设置
void SaveData(double den,int a ,int b);
void DropletSplashSaveDrop();
void SaveTask();


int  GetMyTickCount();
int  TimeInterval();
void DeviceQuery();
void CudaInitialize();
void CudaFree();


//*************************************************************************************************
void ContactAngle(bool side, int tier);
void ComputeEnergy();
void Computetangential_velocity();
bool Cmp(const pair<double, double>& a, const pair<double, double>& b);
void DimConversion(double Length, double MacroLength, double Density, double MacroDensity, double Viscosity, double MacroViscosity);
double EosPressure(const double Density);
void GetThickness(const double Dens[], double & Width);
//double Sigma_IntegralMethod();

//*************************************************************************************************

dim3  DimBlock((DX*DY*DZ + 64 - 1) / 64); dim3 DimThread(64);
//dim3  DimBlock((DX*DY*DZ + 32 - 1) / 32); dim3 DimThread(32);
#define GridIndex   const int I = blockIdx.x*blockDim.x + threadIdx.x;
#define LineIndex   const int i = I / DY / DZ,  j = (I / DZ) % DY,  k = I % DZ;

//*************************************************************************************************

void Initialize()
{
	MModel = MP_CPPRW;
	Tau = 0.7; //Tau = 0.575;
	Tr = 0.68;
	K = 0.2;	//K = 0.1;	//计算三维液滴时取0.2;1
	Ka = 0.001;
	K1 = 0; 
	K2 = -K1;
	BasePt = 0.00;//-0.06(30.572°)   -0.04(49.094°)     0.02(91.975°)   0.072(120.107°)    0.082(124.678°)    0.112(137.039°)   0.132(143.795°)   0.15(149.187°)   0.18(160.378°)
	BasePt2 = -0.08;//-0.08;
	Radius = 60;
	Width = 10;
	//Diameter = 0;// 0.05;//0.2;//cm//之前是0.025cm// +No*0.005;//cm //10微升液体半径大约0.25mm
	Diameter = 0.0001; //0.0001 cm  = 1um
	DropVy = -0.06;// -0.06;// -0.06;
	//TDen = new double[DY][DZ];//临时密度用于计算接触角 Y-Z平面
	TDen = new double[DX][DZ];//临时密度用于计算接触角 X-Z平面

	ReTau = double(1) / Tau;
	Viscosity = (Tau * 2 - 1) / 6;
	DimConversion(10 , Diameter, 1.0, 1.0, Viscosity, 0.01);  // 10latices : 1um (cm) // 1latices den : 1 g/cm^3 
	Gravity = (Diameter == 0) ? 0 : -980 / DimA;
	//量纲变换,求出重力加速度;
	//cout << DimLen<<"	"<<DimTime <<"	"<< Gravity << endl;//Radius=30 Diameter=0.015 Tau=0.7 DimLen=0.00025 DimTime=4.16667e-7

	TasKNum = 1;
	ShowStep = 1;
	SaveStep = 500;
	//AllStep = 100 * 100;
	AllStep = 10 * 500;
	NowStep = StepTime = 0;
	DropStep = 500;//7000
	BeginTime = LastTime = GetMyTickCount();

	//fill((double*)TDen, (double*)TDen + DX * 4, 0);

	Ts[0] = Ts[3] = Ts[5] = Ts[7] = Ts[10] = Ts[12] = Ts[16] = Ts[17] = Ts[18] = 1.0;  //Ammar_2017_JCP;
	Ts[1] = 1;
    	Ts[2] = Ts[4] = Ts[6] = Ts[8] = 1.1;
	Ts[9] = Ts[11] = Ts[13] = Ts[14] = Ts[15] = 1.0 / Tau;  //*/

															/*Ts[0] = Ts[3] = Ts[5] = Ts[7] = 1.0;	Ts[1] = 1.19;
															Ts[2] = Ts[10] = Ts[12] = 1.4; 	Ts[4] = Ts[6] = Ts[8] = 1.2;
															Ts[16] = Ts[17] = Ts[18] = 1.98;
															Ts[9] = Ts[11] = Ts[13] = Ts[14] = Ts[15] = 1.0/Tau;  //*/
}


//*************************************************************************************************
__inline__ __device__ double Feq(int f, double Density, const double Vx, const double Vy, const double Vz)
{
	double DotMet = Vx*Ex[f] + Vy*Ey[f] + Vz*Ez[f];
	return Density * Alpha[f] * (1.0 + 3.0*DotMet + 4.5*DotMet*DotMet - 1.5*(Vx*Vx + Vy*Vy + Vz*Vz));
}




//*************************************************************************************************
__global__ void SetFlowField(char* Type, double* Dens, double* Pote, double* Dist, double* Temp, double * Te )
{
		GridIndex;  LineIndex; if(I >= DXYZ) return;

#ifdef CONTACTANGLE
		if (k<3 || k>DZ - 4)
		{
			Type[I] = SOLID;
			Dens[I] = Pote[I] = _BasePt;
		}
		else 
		{
			Type[I] = FLUID;
			Dens[I] = Pote[I] = 0;
			DVector Vel;   Vel.x = Vel.y = Vel.z = 0;

			double r = sqrtf(Sq(D(i) - DX / 2) + Sq(D(j) - DY / 2) + Sq(D(k) - (_Radius + 2) )); // 液滴中心高度为Radius-20

			Dens[I] = (_DenL + _DenG) / 2 - (_DenL - _DenG) / 2 * tanh(D(r - _Radius) * 2 / _Width);
		}
#elif defined(DROPLETSPLASH)
		if (k<3 || k>DZ - 4)
		{
			Type[I] = SOLID;
			Dens[I] = Pote[I] = 0;
		}
		else 
		{
			Type[I] = FLUID;
			Dens[I] = Pote[I] = 0;
			DVector Vel;   Vel.x = Vel.y = Vel.z = 0;

			double Radius = 50, Width = 10, r = sqrtf(Sq(D(i) - DX / 2) + Sq(D(j) - DY / 2) + Sq(D(k) - DZ / 2));
			Dens[I] = _DenL +(_DenL-_DenG)/2 * (tanh(D(Radius-r)*2/Width) + tanh(D(32-k)*2/Width));		//液滴+液膜;
		}
#elif defined(MEMBRANE)
		if (k<3) //设置底板; 
		{
			Type[I] = SOLID;
			Dens[I] = 0;
			// Pote[I] = _HydroPhobicPt;  //疏水
			// Pote[I] = _HydroPhilicPt;  //亲水
			Pote[I] = 0.010725;  //疏水
		}
		else if ( k >DZ - 4) //设置顶板;
		{
			Type[I] = SOLID;
			Dens[I] = 0;
			// Pote[I] = _HydroPhilicPt;  //亲水
			// Pote[I] = _HydroPhobicPt;  //疏水
			// Pote[I] = 0.010725;  //90
			Pote[I] = -0.04;  //50
		}
		else if ( k >= _PoreBottom && k <= _PoreTop)		//设置膜和孔
		{
			if(Sq(D(i) - _PoreCenterX) + Sq(D(j) - _PoreCenterY) < Sq(_PoreRadius) ) 
			{
				Type[I] = FLUID;
				Pote[I] = 0;
			}
			else
			{
				Type[I] = SOLID;
				Dens[I] = 0;
				// Pote[I] = _HydroPhilicPt;
				// Pote[I] = _HydroPhobicPt;
				Pote[I] = 0.010725;
			}
		}
		else 						//设置液体
		{
			Type[I] = FLUID;
			Pote[I] = 0;
		}

		DVector Vel;   Vel.x = Vel.y = Vel.z = 0;
		if(Type[I] == FLUID)
		{
			double k1 = D(k) - _HotLiquidBottom, k2 = D(k) - _HotLiquidTop;
			double k3 = D(k) - _ColdLiquidBottom, k4 = D(k) - _ColdLiquidTop;

			Dens[I] = _DenG + (_DenL-_DenG)/2 * (tanh(k1 * 2 / _Width) - tanh(k2 * 2 / _Width) + tanh(k3 * 2 / _Width) - tanh(k4 * 2 / _Width));
			// Dens[I] = _DenG + (_DenL-_DenG)/2 * (tanh(k1 * 2 / _Width) - tanh(k2 * 2 / _Width));
			// Dens[I] = _DenG + (_DenL-_DenG)/2 * (tanh(k3 * 2 / _Width) - tanh(k4 * 2 / _Width));
		}

		//温度场初始化;
		Te[I] = _Thot;  //全流场恒温实验 

		// if(k <= _PoreTop)  Te[I] = _Thot;		//以膜上表面为界，上下形成温差
		// else Te[I] = _Tcold;
#endif 

	for (int f = 0; f<DQ; ++f)
	{
		Dist[f*DXYZ + I] = Feq(f, Dens[I],0,0,0);
	}
	
}


//*************************************************************************************************
__device__ void LocalCollideMrt(double * Dens, double * Dist, const double Vx, const double Vy, const double Vz, int I, int i, int j, int k)
{
	double Den = Dens[I];
	double Meq[DQ], Mf[DQ], Mc[DQ];

	double Qp[19] = { 0 };//
		
	//Define_ijk5;
	//Qp[1] = -19 * (_K1 + 3 * _K2) * (GradX5(Dens)* GradX5(Dens) + GradY5(Dens) * GradY5(Dens) + GradZ5(Dens) * GradZ5(Dens));
	//Qp[9] = -_K1 * (2 * GradX5(Dens) * GradX5(Dens) - GradY5(Dens) * GradY5(Dens) - GradZ5(Dens) * GradZ5(Dens));
	//Qp[11] = -_K1 * (GradY5(Dens) * GradY5(Dens) - GradZ5(Dens) * GradZ5(Dens));
	//Qp[13] = -_K1 * GradX5(Dens) * GradY5(Dens);
	//Qp[14] = -_K1 * GradZ5(Dens) * GradY5(Dens);
	//Qp[15] = -_K1 * GradX5(Dens) * GradZ5(Dens);
	
	Define_ijk7;
	Qp[1] = -19 * (_K1 + 3 * _K2) * (Gradx7(Dens) * Gradx7(Dens) + Grady7(Dens) * Grady7(Dens) + Gradz7(Dens) * Gradz7(Dens));
	Qp[9] = -_K1 * (2 * Gradx7(Dens) * Gradx7(Dens) - Grady7(Dens) * Grady7(Dens) - Gradz7(Dens) * Gradz7(Dens));
	Qp[11] = -_K1 * (Grady7(Dens) * Grady7(Dens) - Gradz7(Dens) * Gradz7(Dens));
	Qp[13] = -_K1 * Gradx7(Dens) * Grady7(Dens);
	Qp[14] = -_K1 * Gradz7(Dens) * Grady7(Dens);
	Qp[15] = -_K1 * Gradx7(Dens) * Gradz7(Dens);

	// Ammar_2017_JCP
	Meq[0] = Den; 													//rho
	Meq[1] = Den * (-11.0 + 19.0*(Vx*Vx + Vy*Vy + Vz*Vz));			//e
	Meq[2] = Den * (3.0 - 11.0 / 2 * (Vx*Vx + Vy*Vy + Vz*Vz));		//epsilon
	Meq[3] = Den * Vx;									// j_x
	Meq[4] = Den * Vx * -2 / 3;							// q_x
	Meq[5] = Den * Vy;									// j_y	
	Meq[6] = Den * Vy * -2 / 3;							// q_y	
	Meq[7] = Den * Vz; 									// j_z
	Meq[8] = Den * Vz * -2 / 3;							// q_z
	Meq[9] = Den * (Vx*Vx * 2 - Vy*Vy - Vz*Vz);			//	3p_xx
	Meq[10] = Den * (Vx*Vx * 2 - Vy*Vy - Vz*Vz) / -2; 	// 3PI_xx
	Meq[11] = Den * (Vy*Vy - Vz*Vz);					// p_ww
	Meq[12] = Den * (Vy*Vy - Vz*Vz) / -2;				// PI_ww
	Meq[13] = Den * Vx * Vy;							//p_xy
	Meq[14] = Den * Vy * Vz;							//p_yz
	Meq[15] = Den * Vx * Vz;							//p_xz
	Meq[16] = 0;										//\phi _x
	Meq[17] = 0;										//\phi _y
	Meq[18] = 0;										//\phi _z

	//Lallemand 2003  D3Q19  with T evolution
	// Meq[0] = Den; 													//rho
	// Meq[1] = Den * (-11.0 + 19.0*(Vx*Vx + Vy*Vy + Vz*Vz));			//e
	// Meq[1] = Den * 57 * (1.0/3 - 10.0/19) + 57.0/2(5.0/3 - \gamma ) * (Vx*Vx + Vy*Vy + Vz*Vz) + 57.0 * Te;			//e
	// Meq[2] = Den * (3.0 - 11.0 / 2 * (Vx*Vx + Vy*Vy + Vz*Vz));		//epsilon
	// Meq[3] = Den * Vx;									// j_x
	// Meq[4] = Den * Vx * -2 / 3;							// q_x
	// Meq[5] = Den * Vy;									// j_y	
	// Meq[6] = Den * Vy * -2 / 3;							// q_y	
	// Meq[7] = Den * Vz; 									// j_z
	// Meq[8] = Den * Vz * -2 / 3;							// q_z
	// Meq[9] = Den * (Vx*Vx * 2 - Vy*Vy - Vz*Vz);			//	3p_xx
	// Meq[10] = 0; 										// 3PI_xx
	// Meq[11] = Den * (Vy*Vy - Vz*Vz);					// p_ww
	// Meq[12] = 0;										// PI_ww			*
	// Meq[13] = Den * Vx * Vy;							//p_xy
	// Meq[14] = Den * Vy * Vz;							//p_yz
	// Meq[15] = Den * Vx * Vz;							//p_xz
	// Meq[16] = 0;										//\phi _x
	// Meq[17] = 0;										//\phi _y
	// Meq[18] = 0;										//\phi _z

	// //Vy = 0 equilibrium moments
	// Meq[0] = Den; 													//rho
	// Meq[1] = Den * (-11.0 + 19.0*(Vx*Vx + Vz*Vz));			//e
    // // Meq[1] = Den * (-2.0 + 3.0 * (Sq(Vx) + Sq(Vy)));  
	// Meq[2] = Den * (3.0 - 11.0 / 2 * (Vx*Vx + Vz*Vz));		//epsilon
	// // Meq[2] = Den * (1.00 - 3.0 * (Sq(Vx) + Sq(Vy)));  
	// Meq[3] = Den * Vx;									// j_x
	// Meq[4] = Den * Vx * -2 / 3;							// q_x
    // // Meq[4] = Den * Vx * -1;     
	// Meq[5] = 0;//Den * Vy;									// j_y	
	// Meq[6] = 0;//Den * Vy * -2 / 3;							// q_y	
	// Meq[7] = Den * Vz; 									// j_z
	// Meq[8] = Den * Vz * -2 / 3;							// q_z
    // // Meq[8] = Den * Vz * -1;     
	// Meq[9] = Den * (Vx*Vx - Vz*Vz);			//	3p_xx
	// Meq[10] = Den * (Vx*Vx * 2 - Vz*Vz) / -2; 	// 3PI_xx
	// Meq[11] = Den * (0 - Vz*Vz);					// p_ww
	// Meq[12] = Den * (0 - Vz*Vz) / -2;				// PI_ww
	// Meq[13] = 0;//Den * Vx * Vy;							//p_xy
	// Meq[14] = 0;//Den * Vy * Vz;							//p_yz
	// Meq[15] = Den * Vx * Vz;							//p_xz
	// Meq[16] = 0;										//\phi _x
	// Meq[17] = 0;										//\phi _y
	// Meq[18] = 0;										//\phi _z


	//D2Q9
    // Meq[0] = t.Den;             //rho
    // Meq[1] = t.Den * (-2.0 + 3.0 * (Sq(t.Vx) + Sq(t.Vy)));  //e
    // Meq[2] = t.Den * (1.00 - 3.0 * (Sq(t.Vx) + Sq(t.Vy)));  //epsilon
    // Meq[3] = t.Den * t.Vx;          //jx
    // Meq[4] = t.Den * t.Vx * -1;     //qx
    // Meq[5] = t.Den * t.Vy;          //jy
    // Meq[6] = t.Den * t.Vy * -1;     //qy
    // Meq[7] = t.Den * (Sq(t.Vx) - Sq(t.Vy));     //p_xx
    // Meq[8] = t.Den * t.Vx * t.Vy;               //p_xy 

	for (int f = 0; f<DQ; ++f)	//convert into the momentum space;
	{
		Mf[f] = 0;
		for (int i = 0; i < DQ; ++i)	Mf[f] += M[f][i] * Dist[DXYZ*i + I];
	}

	for (int f = 0; f<DQ; ++f)	//collide in the momentum sapce;
	{
		Mf[f] = Mf[f] - _Ts[f] * (Mf[f] - Meq[f]) + Qp[f] * _Ts[f]; // +Mc[f];
	}

	for (int f = 0; f<DQ; ++f)  //convert back to the lattice space;
	{
		Dist[DXYZ*f + I] = 0;
		for (int i = 0; i < DQ; ++i)  Dist[DXYZ*f + I] += R[f][i] * Mf[i];
	}
}   //*/


//*************************************************************************************************
#define DF(x)  Dist[(x)*DXYZ+I]
__global__ void GlobalCollide(char* Type, double* Dens, double* Pote, double* Dist, double* Temp, double *MVx, double *MVy, double *MVz, double *Vx, double *Vy, double*Vz, double *Fx, double * Fy, double *Fz)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	

	if (Type[I] == FLUID)
	{

		//计算宏观速度;
		MVx[I] = Vx[I] + Fx[I] / Dens[I] * 0.5;
		MVy[I] = Vy[I] + Fy[I] / Dens[I] * 0.5;
		MVz[I] = Vz[I] + Fz[I] / Dens[I] * 0.5;
		//double Melo = sqrt(Sq(MVx) + Sq(MVy) + Sq(MVz));   //宏观速度的模; //*/

		//计算平衡速度;
		double EVx = Vx[I] + Fx[I] / Dens[I];
		double EVy = Vy[I] + Fy[I] / Dens[I];
		double EVz = Vz[I] + Fz[I] / Dens[I];

		//多弛豫碰撞;
		LocalCollideMrt(Dens, Dist, Vx[I], Vy[I], Vz[I], I, i, j, k);

		for (int f = 0; f<DQ; ++f)
		{
			int ii = i + Ex[f];   if (ii < 0) ii += DX;  else if (ii >= DX) ii -= DX;
			int jj = j + Ey[f];   if (jj < 0) jj += DY;  else if (jj >= DY) jj -= DY;
			int kk = k + Ez[f];   if (kk < 0) kk += DZ;  else if (kk >= DZ) kk -= DZ;

			//Dist[f*DXYZ + I] -= 1./_Tau * (Dist[f*DXYZ + I] - Feq(f, Dens[I], Velo[I]));   //单弛豫模型;

			double Df = Dist[f*DXYZ + I] + Feq(f, Dens[I], EVx, EVy, EVz) - Feq(f, Dens[I], Vx[I], Vy[I], Vz[I]);  //精确差分力项;

			(Type[I(ii, jj, kk)] == FLUID ? Temp[f*DXYZ + I(ii, jj, kk)] : Temp[Re[f] * DXYZ + I]) = Df;  //流动和半程反弹;	
		}
	}
}


//*************************************************************************************************
__global__ void MacroCalculate(char* Type, double* Dens, double* Dist, double* Vx, double *Vy, double*Vz)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if (Type[I] == FLUID)
	{
		Dens[I] = 0;
		for (int f = 0; f<DQ; ++f)
		{
			Dens[I] += Dist[f*DXYZ + I];
		}

		//计算格子速度;
		Vx[I] = (DF(1) + DF(7) + DF(9) + DF(11) + DF(13) - DF(2) - DF(8) - DF(10) - DF(12) - DF(14)) / Dens[I];
		Vy[I] = (DF(3) + DF(7) + DF(8) + DF(15) + DF(17) - DF(4) - DF(9) - DF(10) - DF(16) - DF(18)) / Dens[I];
		Vz[I] = (DF(5) + DF(11) + DF(12) + DF(15) + DF(16) - DF(6) - DF(13) - DF(14) - DF(17) - DF(18)) / Dens[I];
		//if (Dens[I] != Dens[I] || Dens[I] < 0 || Dens[I]>11)
		//{
		//printf("Density:  (%d,%d,%d)   %f     %f\n", i, j, k, Dens[I], Dens[I+1]);
		//}
	}
}


//*************************************************************************************************
__global__ void ChemBoundary(char * Type, double * Dens, double * Pote)
{
	const int i = blockIdx.x, j = threadIdx.x;   const int I = i*DY*DZ + j*DZ;
	const int i1 = (i>0 ? i - 1 : DX - 1), i2 = (i<DX - 1 ? i + 1 : 0), j1 = (j>0 ? j - 1 : DY - 1), j2 = (j<DY - 1 ? j + 1 : 0);

	int kk = 3;
	Dens[I] = Dens[I + 1] = Dens[I+2] = (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;

	kk = DZ - 4;
	Dens[I+DZ-3] = Dens[I + DZ - 1] = Dens[I + DZ - 2] = (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;

	Pote[I] = Pote[I + 1] = Pote[I + 2] = Pote[I + DZ -3] = Pote[I + DZ - 2] = Pote[I + DZ - 1] = _BasePt;

#ifdef COMPLEXCHEMBOUNDARY

	kk = 2;
	Dens[I + 1] = (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;
	kk = DZ - 3;
	Dens[I + DZ - 2] = (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;

	kk = 1;
	Dens[I] =  (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;
	kk = DZ - 2;
	Dens[I + DZ - 1] = (Dens[I(i, j, kk)] * 2 + Dens[I(i1, j1, kk)] + Dens[I(i1, j2, kk)] + Dens[I(i2, j1, kk)] + Dens[I(i2, j2, kk)]) / 6;
#endif
}

__global__ void ChemBoundaryComplex(char * Type, double * Dens, char CalcTargetLayerTag , char SourceDataLayerTag) 
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
    if(Type[I] == CalcTargetLayerTag)
    {
        double avg_den = 0;
        double w = 0;

        for (int f = 1; f < DQ; ++f) 
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == SourceDataLayerTag)
            {
               avg_den += Alpha[f] * Dens[pp];
               w += Alpha[f];
            }
        }
        Dens[I] = avg_den / w;
    }
}

__global__ void ChemBoundaryTag(char *Type , char originalPointType, char nextPointType)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
    if (Type[I] == SOLID)
    {
        for (int f = 1; f < DQ; ++f)
        {
            if(i + Ex[f] >= DX || i + Ex[f] < 0 || j + Ey[f] >= DY || j + Ey[f] < 0 || k + Ez[f] >= DZ || k + Ez[f] < 0)  continue;

			int II = I(i + Ex[f], j + Ey[f], k + Ez[f]);
            if (Type[II] == nextPointType)
            {
					Type[I] = originalPointType;
					break;
            }
        }
    }
}

__global__ void SetPorePote(char *Type, double *Pote)
{
	GridIndex;  LineIndex; 
	if(I >= DXYZ) return;
	if(Type[I] == FLUID || Type[I] == SOLID) return;

	//孔道内部 以及 孔口半径R内的区域 的化学势
	if(Sq(D(i) - DX / 2) + Sq(D(j) - DY / 2) < Sq(_PtRadius))
	{
		
		if(k >= _PoreBottom && k <= _PoreTop - 3)
		{
			// Pote[I] =  _HydroPhobicPt;
			Pote[I] =  0.02; // 97degree
			Pote[I] = 0.035; // 109degree
		}
	}
}

//*************************************************************************************************
__global__ void NonidealForce(char* Type, double* Dens, double* Pote, double* Fx, double* Fy, double* Fz)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if (Type[I] == FLUID)
	{
		Define_ijk7;
		Fx[I] = -Gradx7(Pote) * Dens[I] + Gradx7(Dens) / 3;
		Fy[I] = -Grady7(Pote) * Dens[I] + Grady7(Dens) / 3;
		Fz[I] = -Gradz7(Pote) * Dens[I] + Gradz7(Dens) / 3;
	}
}

//*************************************************************************************************
__global__ void ChemPotential(char * Type, double * Dens, double * Pote, double * Te)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if (Type[I] == FLUID)
	{
		double Den = Dens[I], R0 = 9.7, W0 = 1.5;
		if (Den > R0) { Den = R0 + (Den - R0) / (D(1) + (Den - R0)*W0); }

		double T = Te[I] * _Tc;
		// double T = _Thot * _Tc; //恒温

		Pote[I] = T *log(Den / (D(1) - _B*Den)) - _A / (S2 * 2 * _B)*log((S2 - 1 + _B*Den) / (S2 + 1 - _B*Den)) + T / (D(1) - _B*Den) - _A*Den / (D(1) + _B*Den * 2 - Sq(_B*Den));

#ifdef PTCALCFIVEPOINT
		Define_ijk5;
		Pote[I] = Sq(_K) * Pote[I] - _Ka * GradD5(Dens);
#elif defined(PTCALCSEVENPOINT)
		Define_ijk7;
		Pote[I] = Sq(_K) * Pote[I] - _Ka * GradD7(Dens); 
#endif
	}
}

//*************************************************************************************************
__global__ void CompressField(double * Dens, double * Temp)
{
	const int i = blockIdx.x, j = blockIdx.y, k = threadIdx.x;
	const int I = i*LY*LZ + j*LZ + k;   if (I > LXYZ) return;
	const int II = (i*DY*DZ + j*DZ + k) * DR;
	Temp[I] =Dens[II];
}



/**
 * @brief Converts between different units of measurement.
 *
 * This function calculates conversion factors for length, time, mass, and force based on the provided
 * macroscopic and microscopic values for length, density, and viscosity.
 *
 * @param Length Lattice length.
 * @param MacroLength Macroscopic length.
 * @param Density Lattice density.
 * @param MacroDensity Macroscopic density.
 * @param Viscosity Lattice viscosity.
 * @param MacroViscosity Macroscopic viscosity.
 *
 * @example
	DimConversion(10 lattices, 0.0001cm, 1.0, 1.0, Viscosity, 0.01);
	即10格子对应0.0001cm,
	1.0g/cm^3对应1.0格子密度,
	0.01 cm^2/s对应 viscosity格子黏度
	求算单位是cm s g
 * @return void
 */
void DimConversion(double Length, double MacroLength, double Density, double MacroDensity, double Viscosity, double MacroViscosity)
{
	DimLen = MacroLength / Length;
	DimTime = Viscosity / MacroViscosity * Sq(DimLen);
	DimMass = MacroDensity / Density * Sq(DimLen)*DimLen;

	DimF = DimMass * DimLen / Sq(DimTime);
	DimT = DimMass * Sq(DimLen) / Sq(DimTime);
	DimV = DimLen / DimTime;
	DimA = DimLen / Sq(DimTime);
	DimW = 1.0 / DimTime;
	DimB = 1.0 / Sq(DimTime);

	DimEvaporation =  DimLen / DimTime * 36000 ;  // cm / s *36000 = l /(m^2 * h)
	cout << "DimEvaporation: " <<  DimEvaporation << endl;
	DimEvaporation = DimMass / DimLen / DimLen / DimTime * 36000;
	cout << "DimEvaporation: " <<  DimEvaporation << endl;

	//DimEvaporation =  DimMass / (Sq(DimLen) * DimTime) * 36000 ;  // g / (cm^2 * s)  / (1.0 g/cm^3) = cm^3 / (cm^2 * s) =
}

//*************************************************************************************************
void DataShowSave()
{
	
	if (NowStep%ShowStep != 0 && NowStep%SaveStep != 0)   return;
	
	hipMemcpy(HostDens, Dens, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);
	if ((err = hipGetLastError()) != hipSuccess)   cout << "CudaMemcpy: " << (int)err << "   " << hipGetErrorString(err) << endl;

	Mass = 0;
	//for (int n = 0; n<DXYZ; ++n)
	//{
	//	int k = (n % (DY*DZ)) % DZ;
	//	if (k>1 && k<DZ - 2)
	//	{
	//		Mass += HostDens[n];
	//		if (MaxSpeed < HostMelo[n])   MaxSpeed = HostMelo[n];
	//	}
	//} //*/
	MaxSpeed = 0;
	for (int i = 0; i < DX; i++) {
		for (int j = 0; j < DY; j++) {
			for (int k = 2; k < DZ-2; k++) {
				//				cout << MVx[0] << endl;
				double vel = sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]));
				//	if (k>1 && k<DZ-2)
				//		{
				Mass += Dens[I(i, j, k)];
				if (MaxSpeed < vel) MaxSpeed = vel;
				//	if (MaxSpeed < HostMelo[n])   MaxSpeed = HostMelo[n];
				//		}
			}
		}
	}

	if (NowStep > DropStep)
	{
		if (NowStep%SaveStep == 0)
		{
			SaveTask();
			//SaveAllByType(); 
			SaveVel();
		}
	}

	if (NowStep%ShowStep == 0)
	{
		TimeInterval();
		cout << "Laplace: "<<Laplace << endl;
		double We = DropVy * DropVy * 2 * DenL * Radius / Laplace;
		cout << NowStep << "   " << setiosflags(ios::fixed) << setprecision(12) << Mass << "   " << MaxSpeed << "   " << setprecision(3) << Tr << "   " << CaLeft << "   " << CaRight << "   " << CaCap << "   " << We << "   " << StepTime << endl;
	}

	/*if (NowStep%SaveStep == 0)
	{
	ofstream File;
	if( NowStep == SaveStep )
	{
	File.open( "data/CoExistence_PRW.txt" );
	File<<"DenG   Tr   DenL   Tr   Ratio"<<endl;
	File.close();
	}

	File.open( "CoExistence_PRW.txt", ios::app );
	File<<HostDens[I(DX/2,0,DZ/2)]/Rc<<"   "<<Tr<<"   "<< HostDens[I(DX/2,DY/2,DZ/2)]/Rc<<"   "<<Tr<<"   "<< HostDens[I(DX/2,DY/2,DZ/2)]/HostDens[I(DX/2,0,DZ/2)]<<endl;
	File.close();
	}  //*/
}



 void ContactAngle(bool side, int tier)
{
	double PI2 = 3.14159265358979323846264338327950288;
	double MidDen = (DenG + DenL) / 2;
	double Ca = 0;
	double Px[8] = { 0 };
	hipMemcpy(HostDens, Dens, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);
	if (side == 1)
	{
		for (int j = 0; j <= DZ - 1; ++j)for (int i = 0; i < DY ; ++i)
		{
	
			TDen[i][j] = HostDens[I(tier, i, j)];//保存Y-Z平面上的密度
		}
	}
	else
	{
		for (int j = 0; j <= DZ - 1; ++j)  for (int i = 0; i<DX ; ++i)
		{
			TDen[i][j] = HostDens[I(i, tier, j)];//保存X-Z平面上的密度
		}
	}

	char FileName[256];
    sprintf(FileName, "data/XZ_%f.txt" , BasePt);
	ofstream File(FileName);
	File << "i   j   Den" << endl;	
	for (int j = 0; j <= DZ - 1; ++j)  for (int i = 0; i<DX; ++i)
	{
		File << i << ' ' << j << ' ' << TDen[i][j] << endl;//保存X-Z平面上的密度
	}
	File.close();


	/*************************改进插值计算接触角*****************************/
	int begin = 2;//从j=3开始找界面点
				  //寻找液滴的界面

	for (int j = begin; j <= begin + 3; ++j)
	{
		int n = (j - begin) * 2;
		for (int i = 0; i < DX / 2 + 2; ++i)
		{			
			int ii = i + 1;
		int iii = i + 2;
			double &t0 = TDen[i][j], &t1 = TDen[ii][j], &t2 = TDen[iii][j];
			if (t1 < MidDen && t2 >= MidDen)
			{

				Px[n] = D(ii) + (t1 - MidDen) / (t1 - t2);	//第j行左边的气液界面位置;
				break;
			}
		}
		for (int i = DX - 2; i > DX / 2 - 2; --i)
		{
			int ii = i - 1;
			int iii = i - 2;
			double &t0 = TDen[i][j], &t1 = TDen[ii][j], &t2 = TDen[iii][j];
			if (t1 < MidDen && t2 >= MidDen)
			{

				//Px[n + 1] = D(iii) * ((MidDen - t0.Den) * (MidDen - t1.Den)) / ((t2.Den - t0.Den) * (t2.Den - t1.Den)) + D(ii) * ((MidDen - t0.Den) * (MidDen - t2.Den)) / ((t1.Den - t0.Den) * (t1.Den - t2.Den)) + D(i) * ((MidDen - t1.Den) * (MidDen - t2.Den)) / ((t0.Den - t1.Den) * (t0.Den - t2.Den));
				Px[n + 1] = D(ii) - (t1 - MidDen) / (t1 - t2);
				break;
			}
		}
	}
	//Px[0] Px[1]是固体点上的相界面

	//初始化值
	double y0 = 2;// 外推的高度
	double y1 = 3, y2 = 4, y3 = 5;
	//判断第三排是否有点
	if (Px[0] != D(0) && Px[1] != D(0))
	{
		//线性外推
		//左边接触点
		XLeft = Px[2] * ((y0 - y2) / (y1 - y2)) + Px[4] * ((y0 - y1) / (y2 - y1));
		//double x_L = Px[2] * ((y0 - y3) / (y2 - y3)) + Px[4] * ((y0 - y2) / (y3 - y2));
		//右边接触点
		XRight = Px[3] * ((y0 - y2) / (y1 - y2)) + Px[5] * ((y0 - y1) / (y2 - y1));
		// double x_R = Px[3] * ((y0 - y3) / (y2 - y3)) + Px[5] * ((y0 - y2) / (y3 - y2));
		 
		//接触角在175°以下用
		Ca = atan(D(y1 - y0) / (Px[2] - XLeft));
		if (Ca < 0)  Ca += PI2;   Ca = Ca / PI2 * 180;
		CaLeft = Ca;

		if(Ca > 175) cout << " ContactAngle > 175" << endl;

		Ca = atan(-D(y1 - y0) / (Px[3] - XRight));
		if (Ca < 0)  Ca += PI2;   Ca = Ca / PI2 * 180;
		CaRight = Ca;

		if(Ca > 175) cout << " ContactAngle > 175" << endl;
		//if (XLeft < XLeft_min)XLeft_min = XLeft;
		//if (XRight > XRight_max)XRight_max = XRight;

		////从计算液滴水平方向的中心点位置,计算液滴的最高点位置;
		double H, L = XRight - XLeft;

		//
		//for (int i = int((Px[4] + Px[5]) / 2 + 0.5), j = 2; j < DY - 2; ++j)//原 int i = int((Px[2] + Px[3]) / 2 + 0.5)
		for (int i = int((Px[4] + Px[5]) / 2 + 0.5), j = 2; j < DZ - 2; ++j)//原 int i = int((Px[2] + Px[3]) / 2 + 0.5)
		{
			double &t = TDen[i][j], &t1 = TDen[i][j + 1];
			if (t >= MidDen && t1 < MidDen)   H = D(j) + (t - MidDen) / (t - t1) - (y0);//-(y0+1) 因为外推的高度是第一层流体点
		}

		//当存在x_R - x_L为负数时，需要重新计算L，H
		if (L <= 0)
		{
			L = Px[3] - Px[2];
			H = H - 0.5;
		}
		if (XLeft>DY / 2 && XRight<DY / 2)
		{
			XLeft = DY / 2;
			XRight = DY / 2;
		}
		//采用球冠法用公式计算接触角;
		double R = (H * H * 4 + L * L) / H / 8;
		CaCap = atan(L / (R - H) / 2);
		if (CaCap < 0)  CaCap += PI2;   CaCap = CaCap / PI2 * 180;
	}
	else//第三排无值，角度为0，三相接触点在中间
	{
		XLeft =  DY / 2;
		XRight = DY / 2;
		CaLeft = 0;
		CaRight = 0;
	}

	// if (SpreadTime == 0 && XLeft <DY / 2 && XRight > DY / 2)
	// {
	// 	starttime = NowStep;
	// 	SpreadTime = 1;
	// }
	// if (SpreadTime == 1 && CaLeft==0 && CaRight==0)
	// {
	// 	endtime = NowStep + 1;
	// 	SaveSpreadtime();
	// 	SpreadTime = 2;
	// }


	// //找出液滴的质心  Y-Z平面
	// Mx = DX / 2;
	// My = 0;
	// My2 = 0;
	// Mz = 0;
	// Mz2 = 0;
	// Mass = 0;
	// DropMass = 0;
	// for (int j = 0; j <= DZ - 1; ++j)for (int i = 0; i < DY ; ++i)//积分法求质心位置  所有流体点
	// {
	// 	double &t = TDen[i][j];
	// 	Mass += t;
	// 	My += t*i;
	// 	Mz += t*j;

	// }
	// My /= Mass;
	// Mz /= Mass;
	// for (int j = 0; j <= DZ - 1; ++j)for (int i = 0; i < DY; ++i)//积分法求质心位置  所有流体点
	// {
	// 	double &t = TDen[i][j];
	// 	if (t >= (DenG + DenL) / 2)
	// 	{
	// 		DropMass += t;
	// 		My2 += t*i;
	// 		Mz2 += t*j;
	// 	}

	// }
	// My2 /= DropMass;
	// Mz2 /= DropMass;	
}

//计算液滴各个点速度
 void ComputeEnergy()
 {

	 ofstream File;
	 char FileName[256];
	 sprintf(FileName, "data/N4_Energy_or MV_%.3f.txt",BasePt);
	 if (NowStep == 1)
	 {
		 File.open(FileName);
		 File << "NowStep   Radius   Diameter   Tau   DropVy   V   VXY   VZ   MV   MVXY   MVZ   energy   energyXY   energyZ   Time" << endl;
		 File.close();
	 }	 
	 double V = 0;//整体液滴的速度
	 double VXY = 0;//液滴XY方向速度
	 double VZ = 0;//整体液滴的Z方向速度

	 double MV = 0;//整体液滴的动量 
	 double MVXY = 0;//液滴XY方向动量
	 double MVZ = 0;//液滴在Z方向动量

	 double energy = 0;//整体液滴的能量 
	 double energyXY = 0;//液滴XY方向能量
	 double energyZ = 0;//液滴在Z方向能量
	 
	 for (int i = 2; i < DX - 2; i++)
		 for (int j = 2; j < DY - 2; j++)
			 for (int k = 2; k < DZ - 2; k++)
			 {
				 if (Type[I(i, j, k)] == FLUID && Dens[I(i, j, k)]>(DenG+DenL)/2)
				 {
					 V   += sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]));
					 VXY += sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]));
					 VZ  += MVz[I(i, j, k)];

					 MV    += Dens[I(i, j, k)] * sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]));
					 MVXY  += Dens[I(i, j, k)] * sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]));
					 MVZ   += Dens[I(i, j, k)] * MVz[I(i, j, k)];

					 energy   += Dens[I(i, j, k)] * (Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]))/2;
					 energyXY += Dens[I(i, j, k)] * (Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)])) / 2;
					 energyZ  += Dens[I(i, j, k)] * Sq(MVz[I(i, j, k)]) / 2;
				 }
			 }
	 File.open(FileName, ios::app);
	 File << NowStep << "   " << Radius << "   " << Diameter << "   " << Tau << "   " << DropVy << "   " << V << "   " << VXY << "   " << VZ << "   " << MV << "   " << MVXY << "   " << MVZ << "   " << energy << "   " << energyXY << "   " << energyZ << "   " << NowStep*DimTime << endl;
	 File.close();  //*/
 }

 //计算液滴各个点的切向速度
 void Computetangential_velocity()
 {

	 //double alpha = atan(40.11 / 90);//里面的值需要double
	 //if (alpha < 0)  alpha += 3.1415926;   alpha = alpha / 3.1415926 * 180; //必须将弧度转化成角度
	 //cout << alpha << endl;
	 //cout << cos(60.0 / 180 * 3.1415926) << endl;
	 double PI2= 3.14159265358979323846264338327950288;
	 double tangential_vel = 0;		//垂直速度
	 double Mtangential_vel = 0;		//垂直速度动量
	 double mmv = 0;
	 double alpha = 0;				//格子点与水平方向Vx的夹角
	 double beta = 0;				//合速度V与水平速度Vx的夹角
	 double gamma = 0;				//
	 double theta = 0;				//合速度V在垂线速度V⊥的夹角
	 double V = 0;					//合速度
	 double tempV = 0;

	 ofstream File;
	 char FileName[256];
	 sprintf(FileName, "data/0722_N4_tangential_velocity_%.3f.txt", BasePt);
	 if (NowStep == 1)
	 {
		 File.open(FileName);
		 File << "NowStep   Radius   Diameter   Tau   DropVy   tangential_vel   Mtangential_vel   mmv   Time" << endl;
		 File.close();
	 }
	

	 for (int i = 2; i < DX - 2; i++)
		 for (int j = 2; j < DY - 2; j++)
			 for (int k = 2; k < DZ - 2; k++)  //k < DZ - 2
			 {
				 if (Type[I(i, j, k)] == FLUID && Dens[I(i, j, k)]>(DenG + DenL) / 2)
				 {
					 //以前的角度方案
					 //V = sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]));			//	XY平面的XY合速度
					 ////cout <<"	V	"<< V << endl;
					 //if (i < DX / 2 && j < DY / 2)
					 //{						 
						// beta = acos(fabs(MVx[I(i, j, k)]) / V);
						// //cout << "1 	beta	" << beta <<"	i:"<<i << "	j:" <<j << "	k:" <<k<< " 	MVx[I(i, j, k)])	" << MVx[I(i, j, k)] << "	MVy[I(i, j, k)])	" << MVy[I(i, j, k)] << endl;
						// beta = beta / PI2 * 180;									//  弧度转化为角度
						//// cout << "	beta/ PI2 * 180	" << beta << endl;

						// alpha = atan(fabs(D(j - DY / 2) / (i - DX / 2)));
						// //cout << "	alpha	" << alpha << endl;
						// alpha = alpha / PI2 * 180;									//  弧度转化为角度
						//// cout << "	alpha/ PI2 * 180	" << alpha << endl;

						// gamma = fabs(beta + alpha) / 180 * PI2;					//  角度转化为弧度，方便后续计算，可直接用

						// tempV = V * cos(PI2 / 2 - gamma);
					 //}
					 //else
					 //{	
						// if (i == DX / 2)
						// {
						//	 tempV = fabs(MVx[I(i, j, k)]);							//处于圆中线位置切向速度等于Vx
						// }
						// else
						// {
						//	 beta = acos(fabs(MVx[I(i, j, k)]) / V);
						//	 //cout << "2 	beta	" << beta << "	MVx[I(i, j, k)])	" << MVx[I(i, j, k)] << "	MVy[I(i, j, k)])	" << MVy[I(i, j, k)] << endl;
						//	 beta = beta / PI2 * 180;								//  弧度转化为角度
						//	// cout << "2	beta/ PI2 * 180	" << beta << endl;
						//	 alpha = atan(fabs(D(j - DY / 2) / (i - DX / 2)));
						//	 //cout << "2	alpha	" << alpha << endl;
						//	 alpha = alpha / PI2 * 180;								//  弧度转化为角度
						//	 //cout << "2	alpha/ PI2 * 180	" << alpha << endl;

						//	 gamma = fabs(beta - alpha) / 180 * PI2;				//  角度转化为弧度，方便后续计算，可直接用

						//	 tempV = V * cos(PI2 / 2 - gamma);
						// }
					 //}
					 //tangential_vel += tempV;
					 //Mtangential_vel += tempV*Dens[I(i, j, k)];
					 //mmv += tempV*tempV*Dens[I(i, j, k)] / 2;
					// cout <<i<<"	"<<j<< "	" << k <<  "	" << MVx[I(i, j, k)] << "	" << MVy[I(i, j, k)] << "	alpha:	"<< alpha<<"	beta:	"<< beta<<"		tempV:	" << tempV << "		" << endl;
					
					 //坐标变换方案
					 alpha = atan2(j - DY / 2, i - DX / 2) / PI2 * 180;
					 if (alpha < 0)alpha = alpha + 360; alpha = alpha / 180 * PI2;
					 //alpha = atan(fabs(D(j - DY / 2) / (i - DX / 2)));				 
					 //alpha = alpha / PI2 * 180;	//弧度转角度
					 //tangential_vel = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha);

					 //if (i >= DX / 2 && j >= DY / 2)c
					 //{
						// tempV = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha);
					 //}
					 //if (i < DX / 2 && j > DY / 2)
					 //{
						// tempV = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha);//PI2-alpha
					 //}
					 //if (i < DX / 2 && j < DY / 2)
					 //{
						// tempV = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha);//PI2 + alpha
					 //}
					 //if (i > DX / 2 && j < DY / 2)
					 //{
						// tempV = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha); //2 * PI2 - alpha
					 //}
					 tempV = MVx[I(i, j, k)] * sin(alpha) + MVy[I(i, j, k)] * cos(alpha);
					 tangential_vel += tempV;
					 Mtangential_vel += tempV * Dens[I(i, j, k)];
				 }
			 }
	 File.open(FileName, ios::app);
	 File << NowStep << "   " << Radius << "   " << Diameter << "   " << Tau << "   " << DropVy << "   " << tangential_vel << "   " << Mtangential_vel << "   " << mmv << "   " << NowStep*DimTime << endl;
	 File.close();  //*/
 }


void SaveTask()
{

	char prefix[] = "./data";
	#ifdef WIN32
		if (_access(prefix , 0) == -1)	//如果文件夹不存在
			_mkdir(prefix);				//则创建
	#else
		if (access(prefix , 0) == -1)	//如果文件夹不存在
			mkdir(prefix, 0777);				//则创建
	#endif

	dim3  Block(LX, LY, 1), Thread(LZ, 1, 1);
	CompressField << < Block, Thread >> > (Dens, Temp);
	hipMemcpy(HostDens, Temp, sizeof(double)*LXYZ, hipMemcpyDeviceToHost);//压缩用这三行
	CompressField << < Block, Thread >> > (Pote, Temp);
	hipMemcpy(HostPote, Temp, sizeof(double)*LXYZ, hipMemcpyDeviceToHost);//压缩用这三行
	CompressField << < Block, Thread >> > (Te, Temp);
	hipMemcpy(HostTe, Temp, sizeof(double)*LXYZ, hipMemcpyDeviceToHost);//压缩用这三行

	hipDeviceSynchronize();
	// hipMemcpy(HostDens, Dens, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);//不压缩用这
	// hipMemcpy(HostPote, Pote, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);//不压缩用这
	// hipMemcpy(HostTe, Te, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);//不压缩用这

	char FileName[256];
	sprintf(FileName, "data/FlowField_%d", NowStep);
	//sprintf(FileName, "FlowField_%d_Pt", int(BasePt * 100));

	TEC_FILE tec_file(FileName);
	tec_file.Title = "Model3D";
	tec_file.Variables.push_back("i");
	tec_file.Variables.push_back("j");
	tec_file.Variables.push_back("k");
	tec_file.Variables.push_back("Density");
	tec_file.Variables.push_back("Pote");
	tec_file.Variables.push_back("Te");
	tec_file.Zones.push_back(TEC_ZONE(FileName));
	tec_file.Zones[0].Max[0] = LZ;//   压缩保存为LZ
	tec_file.Zones[0].Max[1] = LY;//   压缩保存为LY
	tec_file.Zones[0].Max[2] = LX;//   压缩保存为LX
	// tec_file.Zones[0].Max[0] = DZ;//保存所有DZ  
	// tec_file.Zones[0].Max[1] = DY;//保存所有DY  
	// tec_file.Zones[0].Max[2] = DX;//保存所有DX  
	tec_file.Zones[0].Data.push_back(TEC_DATA(Mxyz[0]));  			//不压缩时，要在 CudaInitalize() 中修改为不压缩
	tec_file.Zones[0].Data.push_back(TEC_DATA(Mxyz[1]));			
	tec_file.Zones[0].Data.push_back(TEC_DATA(Mxyz[2]));
	tec_file.Zones[0].Data.push_back(TEC_DATA(HostDens));
	tec_file.Zones[0].Data.push_back(TEC_DATA(HostPote));
	tec_file.Zones[0].Data.push_back(TEC_DATA(HostTe));
	tec_file.write_plt(1);
}

void DropletSplashSaveDrop()
{
	if (NowStep == DropStep)
	{
		for(int i = 2;i < DX - 2;i++)
			for (int j = 2; j < DY - 2; j++)
				for (int k = 3; k < DZ - 3; k++)
				{
					if (Type[I(i, j, k)] == FLUID)
					{
						double r = sqrtf(Sq(D(i) - DX / 2) + Sq(D(j) - DY / 2) + Sq(D(k) - DZ/2));
						if (r <= 50 && Dens[I(i,j,k)]>(DenG + DenL) / 2) {
							
							//Vz[I(i,j,k)] += DropVy;
							//cout << i << "\t" << j << "\t" << k << "\t" << Vz[I(i, j, k)] << endl;
							//cin.get();
						}
					}
				}
	}

	//if (NowStep == 2000 || NowStep == 200||NowStep == 3500 ||NowStep == 4000 ||NowStep == 4700)
	//{
	//	//cout <<NowStep << endl;
	//	SaveDataOriDen();
	//}

	ofstream File;
	char FileName[256];
	sprintf(FileName, "data/Spread_Drop_%.3f.txt", BasePt);
	if (NowStep == 1)
	{		
		File.open(FileName);
		File << "Droprad   BasePt   NowStep   NowStep*DimTime   CaLeft   CaRight   XLeft   XRight   L   Mx   My   Mz   My2   Mz2" << endl;
		File.close();
	}

	File.open(FileName, ios::app);
	File << Radius << "   " << BasePt << "   " << NowStep << "   " << NowStep*DimTime << "   " << CaLeft << "   " << CaRight << "   " << XLeft << "   " << XRight << "   " << (XRight-XLeft)*DimLen << "   " << Mx << "   " << My*DimLen << "   " << Mz*DimLen << "   " << My2*DimLen << "   " << Mz2*DimLen << endl;
	File.close();  //*/

	//ofstream File;
	//if (No == 0)
	//{
	//	File.open("data/Bounce_Drop.txt");
	//	File << "Droprad   BasePt   CaLeft   CaRight   XLeft_min   XRight_max   L" << endl;
	//	File.close();
	//}

	//File.open("Bounce_Drop.txt", ios::app);
	//File << Radius << "   " << BasePt << "   " << CaLeft << "   " << CaRight << "   " << XLeft_min << "   " << XRight_max << "   " << XRight_max - XLeft_min << endl;
	//File.close();  //*/
	//XLeft_min = DX/2;
	//XRight_max = DX/2;
}
//流场中每一个点的动量 //速度以及密度
void SaveVel()
{
	ofstream File;
	char FileName[256];
	sprintf(FileName, "data/N=4_MVel_%dBasePt_%.3f.txt", NowStep,BasePt);	
	if (No == 0)
	{ 
		File.open(FileName);
		File << "i   j   MV" << endl;
		File.close();
	}
	
	for (int i = 0; i < DX; ++i)
		for (int j = 0; j < DY; ++j)
			//if(Dens[I(i,j,2)]>=(DenG + DenL) / 2)
			{
				File.open(FileName, ios::app);
				File << i << "   " << j << "   " << (Dens[I(i, j, 2)] >= (DenG + DenL) / 2 ? Dens[I(i, j, 2)] * sqrt(MVx[I(i, j, 2)] * MVx[I(i, j, 2)] + MVy[I(i, j, 2)] * MVy[I(i, j, 2)] + MVz[I(i, j, 2)] * MVz[I(i, j, 2)]):0) << endl;//保存动量    + MVz[I(i, j, 2)] * MVz[I(i, j, 2)]
				//File <<i << "   " << j << "   " << sqrt(MVx[I(i, j, 2)] * MVx[I(i, j, 2)] + MVy[I(i, j, 2)] * MVy[I(i, j, 2)] + MVz[I(i, j, 2)] * MVz[I(i, j, 2)]) << endl;//保存速度
				File.close();
			}
}

//流场中每一个X-Z平面的Y速度 //速度以及密度
void SaveY_Vel()
{
	ofstream File;
	char FileName[256];
	sprintf(FileName, "data/N=4_Yvel_XZ_%dBasePt_%.3f.dat", NowStep, BasePt);
	if (No == 0)
	{
		File.open(FileName);
		File << "TITLE = Droplet" << endl;
		File << "VARIABLES =  X,   Y,   Z,   Uy" << endl;
		File << "ZONE  I=" << DX << "  Y=" << DY << "  Z=" << DZ << "  F=POINT " << endl;
		File.close();
	}

	FOR_iDX_jDY_kDZ
	{
		File.open(FileName, ios::app);
		File << i << "   " << j << "   " << k << "   " << (Dens[I(i, j, k)] >= (DenG + DenL) / 2 ? MVy[I(i, j, k)]*DimV :-2)<< endl;//保存动量    + MVz[I(i, j, 2)] * MVz[I(i, j, 2)]
																																			   //File <<i << "   " << j << "   " << sqrt(MVx[I(i, j, 2)] * MVx[I(i, j, 2)] + MVy[I(i, j, 2)] * MVy[I(i, j, 2)] + MVz[I(i, j, 2)] * MVz[I(i, j, 2)]) << endl;//保存速度
		File.close();
	}
}

void SaveAllByType()
{
	//hipMemcpy(Type, _Type, sizeof(int)*DXYZ, hipMemcpyDeviceToHost);

	int count = DXYZ;
	double *ai = new double[count];
	double *aj = new double[count];
	double *ak = new double[count];
	double *aden = new double[count];
	double *Ux = new double[count];
	double *Uy = new double[count];
	double *Uz = new double[count];
	for (int i = 0; i < DX; ++i)
	{
		for (int j = 0; j < DY; ++j)
		{
			for (int k = 0; k < DZ; ++k)
			{
				int p = i*DY*DZ + j*DZ + k;
				int xyz = i + j*DX + k*DY*DX;
				ai[xyz] = i;
				aj[xyz] = j;
				ak[xyz] = k;
				aden[xyz] = Dens[I(i, j, k)];
				Ux[xyz] = (Dens[I(i, j, k)] >= (DenG + DenL) / 2 ? MVx[I(i, j, k)] * DimV : -2);				
				Uy[xyz] = (Dens[I(i, j, k)] >= (DenG + DenL) / 2 ? MVy[I(i, j, k)] * DimV : -2);
				Uz[xyz]= (Dens[I(i, j, k)] >= (DenG + DenL) / 2 ? MVz[I(i, j, k)] * DimV : -2);
			}
		}
	}
	char FileName[128];
	sprintf(FileName, "Field_3D_Nowstep=%d", NowStep);
	TEC_FILE tec_file(FileName);
	tec_file.Title = "Field_3D";
	tec_file.Variables.push_back("i");
	tec_file.Variables.push_back("j");
	tec_file.Variables.push_back("k");
	tec_file.Variables.push_back("Ux");
	tec_file.Variables.push_back("Uy");
	tec_file.Variables.push_back("Uz");
	tec_file.Variables.push_back("Density");
	tec_file.Zones.push_back(TEC_ZONE(FileName));
	tec_file.Zones[0].Max[0] = DX;
	tec_file.Zones[0].Max[1] = DY;
	tec_file.Zones[0].Max[2] = DZ;
	tec_file.Zones[0].Data.push_back(TEC_DATA(ai));
	tec_file.Zones[0].Data.push_back(TEC_DATA(aj));
	tec_file.Zones[0].Data.push_back(TEC_DATA(ak));
	tec_file.Zones[0].Data.push_back(TEC_DATA(Ux));
	tec_file.Zones[0].Data.push_back(TEC_DATA(Uy));
	tec_file.Zones[0].Data.push_back(TEC_DATA(Uz));
	tec_file.Zones[0].Data.push_back(TEC_DATA(aden));
	tec_file.write_plt(1);
	delete[] ai;
	delete[] aj;
	delete[] ak;
	delete[] Ux;
	delete[] Uy;
	delete[] Uz;
	delete[] aden;
}


void SavePointVel()
{
	ofstream File;
	char FileName[256];
	sprintf(FileName, "data/PointVel_Diameter%f_BasePt%.3f.txt", Diameter,BasePt);
	if (NowStep == 1)
	{
		File.open(FileName);
		File << "Droprad   NowStep   NowStep*DimTime   Pote   i   j   k   Dens   MVx   MVy   MVz   MVmax" << endl;
		File.close();
	}

	//File.open(FileName, ios::app);
	//cout << 111 << endl;
	for (int i = 0; i < DX; i++)
		for (int j = 0; j < DY; j++)
			
			if (Dens[I(i, j, 2)] >= (DenG + DenL) / 2 && Pote[I(i, j, 1)] == BasePt2 && Dens[I(i+1, j, 2)] < (DenG + DenL) / 2 && Dens[I(i, j+1, 2)] < (DenG + DenL) / 2)    // && Pote[I(i, j, 2)] == BasePt2   
			{
				//cout << 222 << i << "   " << j << "   " << 2 << endl;
				File.open(FileName, ios::app);
				File << Radius << "   " << NowStep << "   " << NowStep*DimTime << "   " << Pote[I(i, j, 1)] << "   " << i << "   " << j << "   " << 2 << "   " << Dens[I(i, j, 2)] << "   " << MVx[I(i, j, 2)] << "   " << MVy[I(i, j, 2)] << "   " << MVz[I(i, j, 2)] << "   " << MaxSpeed << endl;
				break;
				
			}
	File.close();

}

void SaveSpreadtime()
{

	ofstream File;
	if (No == 0)
	{
		File.open("data/time.txt");
		File << "Droprad   Diameter   BasePt   Tau   DropVy   startTime   endtime   spreadtime   vel" << endl;
		File.close();
	}

	double mvel = 0;
	double vel = 0;
	double dropmass = 0;
	for (int i = 0; i < DX; i++) 
	{
		for (int j = 0; j < DY; j++) 
		{
			for (int k = 2; k < DZ - 2; k++)
			{
				if (Dens[I(i, j, k)] > (DenG + DenL) / 2)
				{
					mvel += Dens[I(i, j, k)]* sqrt(Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]));//累加求出液滴的动量
					dropmass += Dens[I(i, j, k)];//累加求出液滴的质量
				}
			}
		}
	}

	vel = mvel / dropmass;//液滴离开瞬间总体的速度
	File.open("time.txt", ios::app);
	File << Radius << "   " <<Diameter << "   " << BasePt << "   " <<Tau<< "   " << DropVy << "   " << starttime << "   " << endtime << "   " << (endtime-starttime)*DimTime << "   " << vel << endl;
	File.close();  //*/
}

void SaveDataOriDen()
{
	//计算表面张力,分别采用拉普拉斯定律和积分法计算;
	double MidPos = DY / 2 - Radius, MidDen = (Dens[I(DX / 2, 0, DZ / 2)] + Dens[I(DX / 2, DY / 2, DZ / 2)]) / 2;
	double TheRad = 0, Integral = 0;
	for (int j = 0; j <= DY / 2; ++j)
	{
		double &t1 = Dens[I(DX/2, j , DZ / 2)], &t2 = Dens[I(DX / 2, j+1, DZ / 2)];
		if (t1 <= MidDen && t2>MidDen)
		{
			MidPos = D(j) + (MidDen - t1) / (t2 - t1);
			TheRad = D(DY / 2) - MidPos;
			break;
		}
	}


	double &t1 = Dens[I(DX / 2, 0, DZ / 2)], &t2 = Dens[I(DX / 2, DY / 2, DZ / 2)];
	//cout << t1 << " " << t2 << " " << endl;
	double Pr0_In = EosPressure(t2);
	double Pr0_Out = EosPressure(t1);
	//cout << Pr0_In << "	" << Pr0_Out << endl;
	Laplace = (K) * (Pr0_In - Pr0_Out) * TheRad / 2;  //拉普拉斯定律求表面张力;


	char TrackName[256];
	ofstream File;
	sprintf(TrackName, "data/3Lacplace_%s_Tau%3.2f_Tr%3.2f.txt", Name(MModel), Tau, Tr);
	if (NowStep == 0)//No==0 NowStep == 501
	{
		File.open(TrackName);
		File << "Tr   Radius   TheRad   1/Rad   DeltaP   Laplace" << endl;
	}
	else
	{
		File.open(TrackName, ios::app);
	}

	File << Tr << "   " << Radius << "   " << TheRad << "   " << 1.0 / TheRad << "   " << Pr0_In - Pr0_Out << "   " << Laplace <<  endl;
	File.close();
}

void Save_interfaceDen()
{	
	//看底部图形origin
	hipMemcpy(HostDens, Dens, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);
	ofstream File;
	if (No == 0)
	{

		File.open("data/interface_Den.txt");
		File << "i   j" << endl;
		File.close();
	}
	for (int i = 0; i < DX - 1; i++)
		for (int j = 0; j < DY - 1; j++)
		{
			if (HostDens[I(i, j, 1)] == -2)
			{
				File.open("interface_Den.txt", ios::app);
				File << i << "   " << j << endl;//<< "   " << HostDens[I(i, j, 1)] 
				File.close();
			}
			
		}
	
	//hipMemcpy(HostDens, Dens, sizeof(double)*DXYZ, hipMemcpyDeviceToHost);
	//char FileName[256];
	//ofstream File;
	//sprintf(FileName, "data/Try_Compare_FlowField_%d.dat", NowStep);
	//File.open(FileName);
	//File << "TITLE = Droplet" << endl;
	//File << "VARIABLES =  X,   Y,   Z,   Density" << endl;
	//File << "ZONE  I=" << DX << "  J=" << DY << "  K=" << DZ << "  F=POINT " << endl;
	////int z = 1;
	//for (int i = 0; i < DX; i++)
	//	for (int j = 0; j < DY ; j++)
	//		for (int k = 0; k < DZ; k++)
	//		{
	//			File << i << ' ' << j << ' ' << k << ' ' << setprecision(6) << HostDens[I(i, j, k)] << endl;
	//			
	//		}
		
}

//*************************************************************************************************
int main(int argc, char *argv[])
{
	DeviceQuery();
	int DeviceNo = * argv[1] - '0';
	hipSetDevice(DeviceNo); //hipSetDevice(1);
	cout << "  Now is running on GPU device " << DeviceNo << endl;

	string TestName = "2D_Film" ;	//3D/2D  
	No = 0;
	
	//for( BasePt = 0.01 ;BasePt >= -0.06; BasePt -= 0.005)
	//for (; No < TasKNum; ++No)
	{
		Initialize();

		setPara(LoadInitFlag, false);	//是否通过加载的方式对流场进行初始化
		// char LoadFileName[] = "checkpoint/CheckPoint_800000Step.txt";	//加载的文件名
		char LoadFileName[] = "";	//加载的文件名
		setPara(SaveCheckPointFlag, true);	//保存CheckPoint

		//自定义setParameter
		setPara(FilmThickness, 50);   //最好设置为偶数
		setPara(HotLiquidThickness, 60);
		setPara(ColdLiquidThickness, 50);
		setPara(Thot, 0.68); setPara(Tr, 0.68);
		setPara(Tcold, 0.62);
		setPara(ShowStep, 10000);
		setPara(AllStep, 20 * 10000);
		setPara(BasePt,0.01);          // 90degree  0.01(Tr0.68)
		setPara(HydroPhilicPt, -0.04);   //亲水 50degree
		//setPara(HydroPhobicPt, 0.07);	//疏水 140degree
		//setPara(HydroPhobicPt, 0.05);	//疏水 120degree
		// setPara(HydroPhobicPt, 0.035);	//疏水 110degree
		// setPara(HydroPhobicPt, 0.02);	//疏水 110degree
		// setPara(HydroPhobicPt, 0.011);
		setPara(HydroPhobicPt, 0.010725);	// 90degree (0.68)

		// setPara()

		SetMultiphase();
		MembraneParaInit();
		CudaInitialize();
		
		if(0 == LoadInitFlag)//以非加载形式对流场进行初始化
		{
			//设置流场，基础的化学势
			SetFlowField << <DimBlock, DimThread >> > (Type, Dens, Pote, Dist, Temp, Te);

			//标记化学势边界点
			ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL1, FLUID); // (type, originPointType, nextPointType)
			ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL2, LEVEL1);
			ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL3, LEVEL2);  //三层

			//设置膜孔的化学势
			SetPorePote << <DimBlock, DimThread >> > (Type, Pote);
		}
		else //通过加载的方式进行初始化			//对流场的设置进一步处理 
		{
			int err_load = LoadCheckPoint(LoadFileName);


			if(err_load == 0)
			{
				cout << "Load CheckPoint Fail!" << endl;
				return 0;
			}
			else
			{
				cout << "Load CheckPoint Success!" << endl;
			}

			// SetHotLiquid<<<DimBlock, DimThread>>>(Type, Dens);
			// AddFilm();
			// SetPlate<<<DimBlock, DimThread>>>(Type, Dens, Pote);
		}

		//显卡内存信息
		printcudaMemoryInfo();

		{
			//保存场的初始化图像
			//origin画初始场
			SaveTask();
			CalcMacroCPU();
			MembraneERCalc();
			plotInitField_origin();		
			ShowData();

			if(LoadInitFlag) 
			{	
				setPara(NowStep,50000);
				AllStep = 30 * 10000;
			}
			else setPara(NowStep,0);

			//演化开始
			for (; NowStep <= AllStep; ++NowStep)
			{
				//dim3  Block(DX, 1, 1), Thread(DY, 1, 1);
				{
					// TemperatureGradient<<<DimBlock, DimThread>>>(Type,  Te, Tx ,  Ty ,  Tz, Td,  MVx,  MVy,  MVz , DVx,  DVy,  DVz ) ;
					// Temperature<<<DimBlock, DimThread>>>(Type,  Dens,  Te,  MVx,  MVy,  MVz , Tx ,  Ty ,  Tz, Td, DVx,  DVy,  DVz ) ;
					//计算化学势边界
					ChemBoundaryComplex<<< DimBlock, DimThread >> > (Type, Dens, LEVEL1, FLUID);
					ChemBoundaryComplex<<< DimBlock, DimThread >> > (Type, Dens, LEVEL2, LEVEL1);	//五点则只需算两层
					ChemBoundaryComplex<<< DimBlock, DimThread >> > (Type, Dens, LEVEL3, LEVEL2);	//计算非理想力时需要三层格点的化学势
				
					ChemPotential << <DimBlock, DimThread >> > (Type, Dens, Pote, Te);
				}

				NonidealForce << <DimBlock, DimThread >> > (Type, Dens, Pote, Fx, Fy, Fz);
				GlobalCollide << <DimBlock, DimThread >> > (Type, Dens, Pote, Dist, Temp, MVx, MVy, MVz, Vx, Vy, Vz, Fx, Fy, Fz);
				double *p = Dist;  Dist = Temp;  Temp = p;
				MacroCalculate << <DimBlock, DimThread >> > (Type, Dens, Dist, Vx, Vy, Vz);
				hipDeviceSynchronize();

				//计算宏观量
				//计算蒸发率
				//显示指标数据
				//保存流场演化过程
				if(NowStep % ShowStep== 0)
				{
					CalcMacroCPU();
					MembraneERCalc();
					ShowData();
					SaveTask();					
				}
			}

			//循环结束
			//保存checkpoint
			if(SaveCheckPointFlag && !err_den && !err_distribution)
			{
				SaveCheckPoint(TestName);
			}

			//压力曲线
			plotEosPressure(TestName);
		}
		CudaFree();
	}

	cout << endl << " Press Enter key to quit ...... ";   cin.get();
	delete[] TDen;
	return 0;
}

//*************************************************************************************************
void SetMultiphase()
{
	Viscosity = (Tau * 2 - 1) / 6;

	switch (MModel)
	{
	case MP_SCVDW: case MP_CPVDW: case MP_P0VDW:
		A = D(9) / 49;   B = D(2) / 21;   Tc = D(4) / 7;     Rc = D(7) / 2;
		break;

	case MP_SCCSE: case MP_CPCSE: case MP_P0CSE:   //注意此处B为除4后的值;
		A = 1.0;       B = 1.0;   Tc = 0.09432870;     Rc = 0.13044388;
		break;

	case MP_SCRKE: case MP_CPRKE: case MP_P0RKE:   //注意此处A为合并alpha系数后的值;
		A = D(2) / 49;   B = D(2) / 21;   Tc = 0.196133;   Rc = 2.729171;
		A *= D(1) / sqrt(Tc*Tr);
		break;

	case MP_SCRKS: case MP_CPRKS: case MP_P0RKS: { //注意此处A为合并alpha系数后的值;
		A = D(2) / 49;   B = D(2) / 21;   Tc = 0.086861;   Rc = 2.729171;  double w = 0.344;
		A *= Sq(D(1) + (0.480 + 1.574*w - 0.176*w*w)*(D(1) - sqrt(Tr)));
	}break;

	case MP_SCPRM: case MP_CPPRM: case MP_P0PRM:
	case MP_SCPRW: case MP_CPPRW: case MP_P0PRW: { //注意此处A为合并alpha系数后的值;
		A = D(2) / 49;  B = D(2) / 21;  Tc = 0.072919;  Rc = 2.65730416;
		double w = (MModel == MP_SCPRW || MModel == MP_CPPRW || MModel == MP_P0PRW) ? 0.344 : 0.011;
		A *= Sq(D(1) + (0.37464 + 1.54226*w - 0.26992*w*w)*(D(1) - sqrt(Tr)));
	}break;
	}

	//check the _Temperature;
	if (Tr < 0.1 && Tr>1)
	{
		cout << "_Temperature is error:  " << Tr << endl;
		return;
	}

	T = Tr * Tc;

	//Read the _Densities of gas and liquid phases;
	string FileName = Name(MModel);
	FileName = "CoCurve/CoCurve_" + FileName.substr(3, 3) + "_2.txt";

	ifstream File(FileName.c_str());
	if (!File.is_open())
	{
		cout << "open file error:  " << FileName << endl;
		return;
	}

	char Buffer[512];
	istringstream Iss;
	DenG = DenL = 0;

	while (!File.eof())
	{
		double T0, DenGas, DenLiquid;
		File.getline(Buffer, 512);
		Iss.clear();  Iss.str(Buffer);
		Iss >> T0 >> DenGas >> DenLiquid;

		if (Eq(T0, Tr))
		{
			DenG = DenGas * Rc;
			DenL = DenLiquid * Rc;
			break;
		}
	}
	File.close();

	if (NowStep == 0)//if (NowStep == 0)
	{
		//SigmaI = Sigma_IntegralMethod();
		double Re = abs(DropVy) * 2 * Radius * 6 / (0.55 * 2 - 1);
		double We = DropVy * DropVy * 2 * DenL * Radius  / Laplace;
		cout << endl << " Multiphase: " << Name(MModel) << "  (" << DX << ", " << DY << ", " << DZ << ")   Tau=" << setprecision(2) << Tau << "   Tr=" << Tr << "   DenL=" << setprecision(5) << DenL << "   Re=" << Re << "   We=" << We << endl;
		cout << "****************************************************************************************************" << endl;
	}
}


//*************************************************************************************************
void CudaInitialize()
{
	hipMallocManaged((void**)&Type, sizeof(char)   *DX*DY*DZ);
	hipMallocManaged((void**)&Dens, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Pote, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Dist, sizeof(double) *DX*DY*DZ*DQ);
	hipMallocManaged((void**)&Temp, sizeof(double) *DX*DY*DZ*DQ);

	hipMallocManaged((void**)&Fx, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&Fy, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&Fz, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&Vx, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&Vy, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&Vz, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&MVx, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&MVy, sizeof(double)  *DX*DY*DZ);
	hipMallocManaged((void**)&MVz, sizeof(double)  *DX*DY*DZ);

	if ((err = hipGetLastError()) != hipSuccess)   cout << "CudaMalloc : " << (int)err << "   " << hipGetErrorString(err) << endl;

	hipMemcpyToSymbol(HIP_SYMBOL(_Ts), &Ts, sizeof(double)*DQ);
	hipMemcpyToSymbol(HIP_SYMBOL(_Tau), &Tau, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_K), &K, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_T), &T, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_A), &A, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_B), &B, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_Ka), &Ka, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_K1), &K1, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_K2), &K2, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_DenL), &DenL, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_DenG), &DenG, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_BasePt), &BasePt, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_BasePt2), &BasePt2, sizeof(double));
	//液滴半径 、直径、重力、Z方向重力
	hipMemcpyToSymbol(HIP_SYMBOL(_Radius), &Radius, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_Width), &Width, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_NowStep), &NowStep, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_DropStep), &DropStep, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_Gravity), &Gravity, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_Tc), &Tc, sizeof(double));
	if ((err = hipGetLastError()) != hipSuccess)   cout << "CudaSymbol : " << (int)err << "   " << hipGetErrorString(err) << endl;

	Mxyz = new short[3][DXYZ];
	HostDens = new double[DXYZ];
	HostMelo = new double[DXYZ];
	HostPote = new double[DXYZ];
	HostTe = new double[DXYZ];


	//压缩时写法
	for (int i = 0; i<LX; ++i)  for (int j = 0; j<LY; ++j)  for (int k = 0; k<LZ; ++k)
	{
		int n = i*LY*LZ + j*LZ + k;
		Mxyz[0][n] = i;
		Mxyz[1][n] = j;
		Mxyz[2][n] = k;
	}

	////不压缩时方法
	//for (int i = 0; i<DX; ++i)  for (int j = 0; j<DY; ++j)  for (int k = 0; k<DZ; ++k)
	//{
	//	int n = i*LY*LZ + j*LZ + k;	
	//	Mxyz[0][I(i, j, k)] = i;
	//	Mxyz[1][I(i, j, k)] = j;
	//	Mxyz[2][I(i, j, k)] = k;
	//}  //*/
}

void CudaFree()
{
	delete[] Mxyz;
	delete[] HostDens;
	delete[] HostMelo;
	delete[] HostPote;
	delete[] HostTe;

	hipFree(Type);
	hipFree(Dist);
	hipFree(Temp);
	hipFree(Dens);
	hipFree(Pote);
	hipFree(Vx);
	hipFree(Vy);
	hipFree(Vz);
	hipFree(MVx);
	hipFree(MVy);
	hipFree(MVz);
	hipFree(Fx);
	hipFree(Fy);
	hipFree(Fz);

	MembraneParaFree(); // 释放膜的参数

	if ((err = hipGetLastError()) != hipSuccess)   cout << "CudaFree : " << (int)err << "   " << hipGetErrorString(err) << endl;
}

//*************************************************************************************************
void DeviceQuery()
{
	//hipDeviceReset();

	int deviceCount = 0, driverVersion = 0;
	err = hipGetDeviceCount(&deviceCount);

	if (err != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)err, hipGetErrorString(err));
		return;
	}

	if (deviceCount == 0)
	{
		printf("There is no available device that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	for (int n = 0; n < deviceCount; ++n)
	{
		hipSetDevice(n);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, n);
		hipDriverGetVersion(&driverVersion);

		printf("Device %d: %s    Ver:%d.%d/%d.%d    Core:%dx%d=%d    Memory:%.3f GB\n", n, \
			deviceProp.name, driverVersion / 1000, (driverVersion % 100) / 10, deviceProp.major, deviceProp.minor,
			deviceProp.multiProcessorCount, _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount,
			(float)deviceProp.totalGlobalMem / 1024 / 1024 / 1024);
	}

	hipSetDevice(0);
	if ((err = hipGetLastError()) != hipSuccess)   cout << "DeviceQuery : " << (int)err << "   " << hipGetErrorString(err) << endl;
}


//*************************************************************************************************
int GetMyTickCount()
{
#ifdef WIN32
	return ::GetTickCount();
#else
	struct timeval tv;	 gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000 + tv.tv_usec) / 1000;
#endif
}
int  TimeInterval()
{
	int NowTime = GetMyTickCount();
	StepTime = NowTime - LastTime;
	LastTime = NowTime;
	return StepTime;
}

bool Cmp(const pair<double, double>& a, const pair<double, double>& b)
{
	return a.second < b.second;
}

void SaveData(double den,int i ,int j)
{
	ofstream File("data/Try_Compare.txt");
	File << "TDen   i   j" << endl;
	File << den << ' ' << i << ' ' << j << endl;
	File.close();
}

//备注：下面的密度和温度需要考虑之前计算时，是否已经折合过，如果折合过则下面密度不用乘Rc,温度同理
double EosPressure(const double Den)
{
	//double Pr0 = -1, Den = Density, TT = T;
	double Pr0 = -1;// Den = Density*Rc, TT = Tr*Tc;
	switch (MModel)
	{
	case MP_CPVDW: Pr0 = T * Den / (D(1) - B*Den) - A*Sq(Den);  break;

	case MP_CPPRW: Pr0 = T * Den / (D(1) - B*Den) - A*Sq(Den) / (D(1) + B*Den * 2 - Sq(B*Den));  break;

	}
	return Pr0;
}

//*************************************************************************************************
//计算界面厚度,由气相到液相提供60个格点数据,界面两侧大约各30个数据;
void GetThickness(const double Dens[], double & Width)
{
	//计算两相界面位置: (1)将格点加密后, 求等摩尔面的位置;  (2)用插值法求半密度点位置;  
	//计算过渡区的宽度: (1)用界面点的密度导数求宽度;        (2)用双曲正切函数拟合最佳宽度;
	//界面位置用等摩尔法求, 理论依据比较好;   用高次拉格朗日插值无法精确求得界面点的导数;
	double DenG = Dens[0], DenL = Dens[60];
	double MidDen = (DenG + DenL) / 2, MidPos = 0;
	for (int i = 0; i <= 60; ++i)
	{
		if (Dens[i] <= MidDen && Dens[i + 1]>MidDen)
		{
			//MidPos = D(i) + Cubic( MidDen, Dens[i-1], Dens[i], Dens[i+1], Dens[i+2] );
			MidPos = D(i) + (MidDen - Dens[i]) / (Dens[i + 1] - Dens[i]);
			break;
		}
	}

	double Diff = 10000;
	for (double W = 0.1; W <= 25; W += 0.01)
	{
		double d = 0;
		for (int i = int(MidPos - 25); i <= int(MidPos + 25 + 1); ++i)
		{
			double Den = (DenG + DenL) / 2 - (DenG - DenL) / 2 * tanh((D(i) - MidPos) * 2 / W);
			d += Sq(Den - Dens[i]);
		}

		if (d < Diff)
		{
			Diff = d;
			Width = W;
		}
	}
}


//double Sigma_IntegralMethod() 
//{
//	GridIndex;  LineIndex;
//
//	double sigma = 0;
//	for (int i = 0; i <= DX / 2; ++i) 
//	{
//		int j = DY / 2, k = DZ / 2;
//		Define_ijk5;
//		sigma += Ka * (GradY5(Dens) * GradY5(Dens) - GradX5(Dens) * GradX5(Dens));
//	}
//	return sigma;
//}
void CalcMacroCPU()
{
    Mass = 0;
    FOR_iDX_jDY_kDZ_Fluid 
    {
		double Den = Dens[I(i, j, k)];
		Mass += Den; //求总质量

		if( Den!=Den || Den<0) 
		{
			err_den = true;
			ofstream errorfile("data/error_Density.txt", ios::app);
			errorfile << NowStep << " " << i << " " << j << " " << k << " " << Den << endl;
			errorfile.close();
		}

		for(int f = 0; f < DQ; ++f) 
		{
			double t = Dist[f*DXYZ + I(i, j, k)];
            if(t <=0 || t >= 5)
			{
				err_distribution = true;
				ofstream errorfile("data/error_distribution.txt", ios::app);
				errorfile << "Dist: "<< NowStep << " " << i << " " << j << " " << k << " " << f << " Dist = " << t << endl;
				errorfile.close();
			}
		}

	}

	if(err_den) 
	{
		cout << " Error Density! " << endl;
		NowStep = AllStep;
	}
	if(err_distribution) 
	{
		cout << " Error Distribution! " << endl;
		NowStep = AllStep;
	}

	// 求最大速度
	MaxSpeed = 0;
	FOR_iDX_jDY_kDZ_Fluid {
		double Mod = Sq(MVx[I(i, j, k)]) + Sq(MVy[I(i, j, k)]) + Sq(MVz[I(i, j, k)]);
		if (Mod > MaxSpeed) {
			MaxSpeed = Mod;
		}
	}
	MaxSpeed = sqrt(MaxSpeed);
}

void ShowData()
{
	//打印表头	
    if (NowStep == 0) 
	{
        cout << "程序参数设置：" << endl;
        cout << "Multiphase: " << Name(MModel) << "    DX=" << DX << "    DY=" << DY << "    DZ=" << DZ << "    Tau=" << Tau << "    Tr=" << Tr << "    DenG=" << DenG << "    DenL=" << DenL << endl;
        cout << "Radius=" << Radius 
            << "    Width=" << Width 
            << "    Ka=" << Ka 
            << "    K=" << K 
            << "    BasePt=" << BasePt 
			<< "  	AllStep=" << AllStep
            << endl;


		//化学势边界计算方式
#ifdef COMPLEXCHEMBOUNDARY
		cout << "化学势边界条件：逐层计算" << endl;
#else
		cout << "化学势边界条件：多层设置为相同" << endl;
#endif


		//化学势计算方式：
		cout << "ChemicalPotential Calculation Method: ";
#ifdef PTCALCFIVEPOINT   //五点
			cout << "FivePoint" << endl;
#elif defined(PTCALCSEVENPOINT)  //七点
			cout << "SevenPoint" << endl;
#else
			cout << "Error:  Please define the Pote calculation method!" << endl;
#endif

		//输出时间戳
		printTimeStamp();
	}	//end of if(NowStep == 0)


#ifdef MEMBRANE
		MembraneParaShow();
#endif

#ifdef CONTACTANGLE	
		ContactAngleShow();
#endif

	if(NowStep == AllStep) printTimeStamp();		//结束时间戳
}



void SaveContactAngle()
{
	ofstream File;
	char FileName[256];
    sprintf(FileName, "data/CP%s_MRT_Tau=%2.1f_FIELD%dx%dx%d_ContactAngle_R%3.1f_Tr%3.1f.txt", Name(MModel) + 3, Tau, DX, DY, DZ, Radius, Tr);
	if(No == 0 && NowStep == 0)
	{		

		File.open(FileName, ios::app); //File << "Droprad   BasePt   NowStep   NowStep*DimTime   CaLeft   CaRight  CaCap  XLeft   XRight   L   Mx   My   Mz   My2   Mz2" << endl; //#
		File << "Droprad" << "  " 
            << "BasePt" << "  "
            << "NowStep" << "  "
            //<< "NowStep*DimTime" << "  "
            << "CaLeft" << "  "
            << "CaRight" << "  "
            << "CaCap" << "  "
            << "XLeft" << "  "
            << "XRight" << "  "
            << endl;
		File.close();
	}

	File.open(FileName, ios::app); //File << Radius << "   " << BasePt << "   " << NowStep << "   " << NowStep*DimTime << "   " << CaLeft << "   " << CaRight << "    " << CaCap << "   " << XLeft << "   " << XRight << "   " << (XRight - XLeft) * DimLength << "   " << Mx << "   " << My * DimLength << "   " << Mz * DimLength << "   " << My2 * DimLength << "   " << Mz2 * DimLength << endl; File.close();  //#
    File << Radius << "  " 
        << BasePt << "  " 
        << NowStep << "  " 
     //   << NowStep * DimTime << "  "
        << CaLeft << "  "
        << CaRight << "  "
        << CaCap << "  "
        << XLeft << "  "
        << XRight << "  "
        << endl;
    File.close(); 
}


void ContactAngleShow()
{
	if(NowStep == 0)
	{
        //液滴的位置
        // cout << "LiquidDrop Position: " 
        //     << "    X=" << DropletPosX 
        //     << "    Y=" << DropletPosY 
        //     << "    Z=" << DropletPosZ 
        //     << endl;
        cout << endl;

        cout << "NowStep" 
        << "    Mass" 
        << "    Den(DX/2 , 0 , 3)" 
        << "    Den(DX/2, DY/2/8, 3)" 
        << "    Den(DX/2, DY/2/4, 3)" 
        << "    MaxSpeed"
        //<< "    Temperature(DX/2, DY/2, DZ*3/4)"
        << "    CaLeft"
        << "    CaRight"
        << "    CaCap"
        << "    XLeft"
        << "    XRight"
        << "    StepTime"  
        << endl;
        cout << "----------------------------------------------------------------------------------------------------" << endl;
	}
    
    cout << setw(9) << NowStep 
    << "    " << setiosflags(ios::fixed) << setprecision(12) << Mass 
    << "    " << setprecision(16) << Dens[I(DX / 2, 0, 3)] /Rc
	<< "    " << Dens[I(DX / 2, DY / 2 / 8, 3)] /Rc
	<< "    " << Dens[I(DX / 2, DY / 2 / 4, 3)] /Rc
    << "    " << setprecision(6)  << MaxSpeed 
    //<< "    " << Grid[DX / 2][DY / 2][DZ * 3/ 4].Te
    << "    " << CaLeft
    << "    " << CaRight
    << "    " << CaCap
    << "    " << XLeft
    << "    " << XRight
    << "    " << TimeInterval()
    << endl;
}



//*****************************************MembraneFunction********************************************************
void MembraneParaInit()
{
	//膜上沿高度
	const int centerZ = DZ / 2;
	PoreTop = centerZ + FilmThickness/2;
	PoreBottom = centerZ - FilmThickness/2;								
	PoreCenterX = DX / 2;
	PoreCenterY = DY / 2;
	PoreRadius = 25;
	HotLiquidTop = PoreBottom;
	HotLiquidBottom = HotLiquidTop - HotLiquidThickness;
	ColdLiquidTop = DZ - 3;
	ColdLiquidBottom = DZ - 3 - ColdLiquidThickness;
	//HydroPhobicPt = 0.08;
	PtRadius = PoreRadius + 10 ;
	
	Porosity = Sq(PoreRadius) / ((DX) * (DY)) * 100 ;

	hipMallocManaged((void**)&DVx, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&DVy, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&DVz, sizeof(double) *DX*DY*DZ);

	hipMallocManaged((void**)&Te, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Tx, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Ty, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Tz, sizeof(double) *DX*DY*DZ);
	hipMallocManaged((void**)&Td, sizeof(double) *DX*DY*DZ);

	if ((err = hipGetLastError()) != hipSuccess)   cout << "MembraneParaInit : " << (int)err << "   " << hipGetErrorString(err) << endl;

	hipMemcpyToSymbol(HIP_SYMBOL(_PoreBottom), &PoreBottom, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_PoreTop), &PoreTop, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_PoreCenterX), &PoreCenterX, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_PoreCenterY), &PoreCenterY, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_PoreRadius), &PoreRadius, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_HotLiquidBottom), &HotLiquidBottom, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_HotLiquidTop), &HotLiquidTop, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_ColdLiquidBottom), &ColdLiquidBottom, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_ColdLiquidTop), &ColdLiquidTop, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_HydroPhobicPt), &HydroPhobicPt, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_HydroPhilicPt), &HydroPhilicPt, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_PtRadius), &PtRadius, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(_Thot), &Thot, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(_Tcold), &Tcold, sizeof(double));
}
void MembraneParaFree()
{
	hipFree(DVx);
	hipFree(DVy);
	hipFree(DVz);
	hipFree(Te);
	hipFree(Tx);
	hipFree(Ty);
	hipFree(Tz);
	hipFree(Td);
}

void MembraneParaShow()
{
	if(NowStep == 0)
	{
		cout <<"Membrane Parameters: " << endl;

		printf("\tPoreBottom: %d  PoreTop: %d  |  PoreCenter (X, Y): %d %d |  PoreRadius: %d\n", PoreBottom, PoreTop, PoreCenterX, PoreCenterY, PoreRadius);
		printf("\tHotLiquidBottom: %d  HotLiquidTop: %d\n", HotLiquidBottom, HotLiquidTop);  
		printf("\tColdLiquidBottom: %d  ColdLiquidTop: %d\n", ColdLiquidBottom, ColdLiquidTop);

		// cout << "PoreBottom: " << PoreBottom 
		// 	<< " PoreTop: " << PoreTop
		// 	<< " PoreCenter (X, Y): " << PoreCenterX << " " << PoreCenterY
		// 	<< " PoreRadius: " << PoreRadius
		// 	<< " HotLiquidBottom: " << HotLiquidBottom
		// 	<< " HotLiquidTop: " << HotLiquidTop
		// 	<< " ColdLiquidBottom: " << ColdLiquidBottom
		// 	<< " ColdLiquidTop: " << ColdLiquidTop
		// 	<< " HydroPhobicPt: " << HydroPhobicPt
		// 	<< " PtRadius: " << PtRadius 
		// 	<< endl;

		cout << "----------------------------------------------------------------------------------------------------" << endl;
		//使得表头和数据对齐
		cout << setw(9) << "NowStep" 
			<< setw(24) << "Mass"
			<< setw(15) << "MaxSpeed"
			<< setw(20) << "NowUpperMass"
			<< setw(20) << "TotalDeltaMass"
			<< setw(15) << "DeltaMass"
			<< setw(15) << "Te[DX/2, DY/2, 225] "
			<< setw(15) << "ER"
			<< setw(15) << "ER_LMH"
			<< setw(15) << "ER_inst"
			<< " 	StepTime"
			<< endl;
	}

	cout << setw(9) << setiosflags(ios::left) << NowStep 
		<< "    " << setw(20) << setiosflags(ios::fixed) << setprecision(12) << Mass
		<< "    " << setw(15) << MaxSpeed
		<< "	" << setw(20)<< setprecision(10) << NowUpperMass
		<< "	" << setw(20)<< setprecision(10) << TotaldeltaMass
		<< "	" << setw(15)<< deltaMass
		<< "	" << setw(10)<< setprecision(10) << Te[I(DX / 2, DY / 2, 225)]
		<< "	" << setw(10)<< setprecision(10) << ER
		<< " 	" << setw(15)<< ER_LMH
		<< " 	" << setw(15)<< ER_inst
		<< "    " << setiosflags(ios::left) << setw(15)<< StepTime
		<< endl;

	//查看压力情况
	// cout << "Pressure(Y = 150) = " << EosPressure(Dens[I(DX / 2, DY / 2, 150)]) << endl;
	// cout << "Pressure(Y = 225) = " << EosPressure(Dens[I(DX / 2, DY / 2, 225)]) << endl;
}

void MembraneSaveData()
{
	//C风格
	// FILE *File;
	// char FileName[256];
	// sprintf(FileName, "data/CP%s_MRT_Tau=%2.1f_FIELD%dx%dx%d_Membrane.txt", Name(MModel) + 3, Tau, DX, DY, DZ);
	// File = fopen(FileName, "a");
	// if(File == NULL)
	// {
	// 	cout << "Open File Error: " << FileName << endl;
	// 	return;
	// }
	// else 
	// {
	// 	fprintf(File, "NowStep   Mass   Den(DX/2 , DY/2, 0)   Den(DX/2, DY/2, DZ/2)   MaxSpeed   StepTime   Evaporation\n");
	// 	fprintf(File, "%d   %f   %f   %f   %f   %d   %f\n", NowStep, Mass, Grid[DX / 2][DY / 2][0].Den, Grid[DX / 2][DY / 2][DZ / 2].Den, MaxSpeed, StepTime, Evaporation);
	// 	fclose(File);
	// }

	//C++风格
	const string folderPath = "data";
	system(string("mkdir -p" + folderPath).c_str());

	string FileName = folderPath 
		+ string(Name(MModel) + 3) 
		+ "_MRT_Tau=" + to_string(Tau) 
		+ "_FIELD" + to_string(DX) + "x" + to_string(DY) + "x" + to_string(DZ) + "_Membrane.txt";
	//如果文件不存在，则创建文件
	ofstream File(FileName, ios::app);
	if(!File.good())
	{
		cout << "ofstream is not good! " <<endl;
		return ;
	}
	if(!File.tellp())
	{
		File << "NowStep" 
			<< "    Mass"
			<< "    MaxSpeed"
			<< "	Thot"
			<< "	Tcold"
			<< "    PoreRadius"
			<< " 	FilmThickness" 	//膜厚度
			<< "    HydroPhobicPt"
			<< "    PtRadius"		//孔口的疏水性半径
			<< " 	Porosity(%)"   // 孔隙率 = （孔隙体积/总体积）* 100%
			<< " 	Viscosity"
			<< "    ER"
			<< "	ER_LMH"
			<< endl;
	}

	File << NowStep 
		<< "    " << setiosflags(ios::fixed) << setprecision(12) << Mass
		<< "    " << setprecision(6) << MaxSpeed
		<< " 	" << Tr
		<< "	" << Tcold
		<< "    " << PoreRadius
		<< "	" << FilmThickness
		<< "    " << HydroPhobicPt
		<< "    " << PtRadius
		<< "    " << Porosity
		<< " 	" << Viscosity
		<< "    " << ER
		<< "	" << ER_LMH
		<< endl;
}

void MembraneERCalc()
{
	NowUpperMass = 0;
	startStep = 0;
	ER = 0;
	ER_inst = 0;
	ER_LMH = 0;

	//const int MetricPos = (PoreTop + ColdLiquidBottom) / 2;
	const int MetricPos = PoreTop;

	FOR_iDX_jDY_kDZ_Fluid
	{

		if(k > MetricPos ) NowUpperMass += Dens[I(i, j, k)];	
	}
	if(NowStep == 0) 
	{
		preStepUpperMass = NowUpperMass;
		startUpperMass = NowUpperMass;
	}
	//if(NowStep == startStep) startUpperMass = NowUpperMass;
	if(NowStep > startStep) 
	{
		TotaldeltaMass = NowUpperMass - startUpperMass;
		ER = TotaldeltaMass / D(NowStep - startStep) / D((DX)*(DY)) ;
	}

	ER_LMH = ER  * DimEvaporation;
	

	if(NowStep >= ShowStep) 
	{

		deltaMass = NowUpperMass - preStepUpperMass;
		ER_inst = deltaMass /D(ShowStep) / D((DX) * (DY)) * DimEvaporation;
	}
	preStepUpperMass = NowUpperMass;

	//绘制蒸发率-时间曲线
	plotER_Time();
}

__global__ void TemperatureGradient(char *Type, double *Te, double *Tx, double *Ty, double *Tz, double *Td, double *MVx, double *MVy, double *MVz, double *DVx, double *DVy, double *DVz)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if (i >= 0 && i <= DX - 1 && j >= 0 && j <= DY - 1 && k >= 0 && k <= DZ - 1) 
	{
		if (k > _PoreTop && Type[I] == FLUID)
		{
			// Define_ijk5;
			// Tx[I] = GradX5(Te);
			// Ty[I] = GradY5(Te);
			// Tz[I] = GradZ5(Te);
			// Td[I] = GradD5(Te);

            // DVx[I] = GradX5(MVx);
            // DVy[I] = GradY5(MVy);
            // DVz[I] = GradZ5(MVz);

			Define_ijk7;
			Tx[I] = GradX7(Te);
			Ty[I] = GradY7(Te);
			Tz[I] = GradZ7(Te);
			Td[I] = GradD7(Te);

            DVx[I] = GradX7(MVx);
            DVy[I] = GradY7(MVy);
            DVz[I] = GradZ7(MVz);
		}
	}
}
__global__ void Temperature(char *Type, double *Den, double *Te, double *MVx, double *MVy, double *MVz ,double *Tx , double *Ty , double *Tz , double *Td, double *DVx, double *DVy, double *DVz ) 
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
    if (i >= 0 && i <= DX - 1 && j >= 0 && j <= DY - 1 && k >= 0 && k <= DZ - 1) 
    {
        // if (k <= _PoreTop)
        // {
        //     Te[I] = _Thot;
        // }
		// if(k > DZ - 4) Te[I] = _Tcold;

        if (k > _PoreTop && Type[I] == FLUID)
        {
			// Define_ijk5;
			//Define_ijk7;
            Te[I] += -(MVx[I] * Tx[I] + MVy[I] * Ty[I] + MVz[I] * Tz[I]) + 0.02 * Td[I] - 0.02 * (DVx[I] +  DVy[I] + DVz[I]); 
        }
    }
}

err_type SaveCheckPoint(const string TestName)
{

	char prefix[] = "./checkpoint";
	string FieldName = "CheckPoint_" + TestName;

	#ifdef WIN32
		if (_access(prefix , 0) == -1)	//如果文件夹不存在
			_mkdir(prefix);				//则创建
	#else
		if (access(prefix , 0) == -1)	//如果文件夹不存在
			mkdir(prefix, 0777);				//则创建
	#endif

	//保存流场信息
	char FileName[256];
	sprintf(FileName, "%s/%s_%dStep.checkpoint", prefix, FieldName.c_str(), (NowStep - 1));
	FILE *File = fopen(FileName, "w"); // FILE *File = fopen("checkpoint/CheckPoint.txt", "w"); //重新写入
	if(File == NULL)
	{
		cout << "CheckPoint.txt open fail!" << endl;
		return 0;
	}

	hipDeviceSynchronize();

	//Type , Dens , Pote, Dist, Temp, Te, V
	fwrite(Type, sizeof(char), DX * DY * DZ, File);
	fwrite(Dens, sizeof(double), DX * DY * DZ, File);
	fwrite(Pote, sizeof(double), DX * DY * DZ, File);
	fwrite(Dist, sizeof(double), DX * DY * DZ * DQ, File);
	fwrite(Te, sizeof(double), DX * DY * DZ, File);
	fwrite(Vx, sizeof(double), DX * DY * DZ, File);
	fwrite(Vy, sizeof(double), DX * DY * DZ, File);
	fwrite(Vz, sizeof(double), DX * DY * DZ, File);

	fclose(File);

	//保存一个流场切片
	sprintf(FileName, "%s/%s_%dStep_Field.txt", prefix, FieldName.c_str(), NowStep - 1);
	ofstream File2(FileName, ios::out);
	if(!File2.good())
	{
		cout << "File open fail!" << endl;
		return 0;
	}
	File2 << "X" << "	" << "Y" << "	" << "Denstity"  << "	" << "Potential" << endl;

	for(int i = 0; i < DX; ++i)
	for(int k = 0; k < DZ; ++k)
	{
		//Solid
		if(Type[I(i, DY/2 ,k)] != FLUID)
			File2 << i << "	" << k << "	" << -1 << "	" << Pote[I(i,DY/2 ,k)] << endl;
		else //Fluid
		{
			File2 << i << "	" << k << "	" << Dens[I(i,DY/2 ,k)]  << "	" << Pote[I(i,DY/2 ,k)] << endl;
		}
	}
	File2.close();

	return 1;
}

err_type LoadCheckPoint(char * FileName)
{

	FILE *File = fopen(FileName, "r");
	if(File == NULL)
	{
		cout << "CheckPoint.txt open fail!" << endl;
		return 0;
	}

	//Type , Dens , Pote, Dist, Temp, Te, V
	fread(Type, sizeof(char), DX * DY * DZ, File);
	fread(Dens, sizeof(double), DX * DY * DZ, File);
	fread(Pote, sizeof(double), DX * DY * DZ, File);
	fread(Dist, sizeof(double), DX * DY * DZ * DQ, File);
	fread(Te, sizeof(double), DX * DY * DZ, File);
	fread(Vx, sizeof(double), DX * DY * DZ, File);
	fread(Vy, sizeof(double), DX * DY * DZ, File);
	fread(Vz, sizeof(double), DX * DY * DZ, File);

	hipDeviceSynchronize();

	fclose(File);

	return 1;
}

//-------------------------------------加载流场后需使用的临时函数----------------------------------------------------------
//当场完全平衡后加入膜的设置
void AddFilm()
{
	SetFilm<<< DimBlock, DimThread>>>(Type, Dens, Pote);

	ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL1, FLUID); // (type, originPointType, nextPointType)
	ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL2, LEVEL1);
	ChemBoundaryTag << <DimBlock, DimThread >> > (Type, LEVEL3, LEVEL2);

	SetPorePote<<< DimBlock, DimThread>>>(Type, Pote);
}

__global__ void SetFilm(char * Type, double * Dens, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k >= _PoreBottom && k <= _PoreTop)		//设置膜和孔
	{
		if(Sq(D(i) - _PoreCenterX) + Sq(D(j) - _PoreCenterY) >= Sq(_PoreRadius) ) 
		{
			Type[I] = SOLID;
			Dens[I] = 0;
			Pote[I] = _BasePt;
		}
	}
}

__global__ void SetHotLiquid(char * Type, double * Dens)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if(Type[I] != FLUID) return;

	double k1 = D(k) - _HotLiquidBottom, k2 = D(k) - _HotLiquidTop;

	if(k >= _HotLiquidBottom - 30 && k <= _HotLiquidTop + 30)		
		//Dens[I] = _DenG + (_DenL-_DenG)/2 * (tanh(k1 * 2 / _Width) - tanh(k2 * 2 / _Width)); 
		Dens[I] = 0.01348 + (8.667 - 0.01348 )/2 * (tanh(k1 * 2 / _Width) - tanh(k2 * 2 / _Width));  //通过上一次演化后的实际测量的密度来设置
}

//修改上下底板的润湿性
__global__ void SetPlate(char * Type, double * Dens, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if(k >= DZ - 3)		Pote[I] = _BasePt;
	if(k <= 2) Pote[I] = _BasePt;
}

//-------------------------------需要绘制的曲线图-----------------------------------------------
void plotER_Time()
{
	ofstream File("data/ER_time.txt", ios::app);
	if(!File.good())
	{
		cout << "File ER.txt open fail!" << endl;
		return;
	}
	if(NowStep == 0)
	{
		File << "TimeStep" << "    EvaporationRate(lattice)" << "    TotalEvaporationRate(Lm^-2h^-1)" << "    InstantEvaporationRate(Lm^-2h^-1)" << endl;
	}

	File << NowStep << "    " << ER << "    " << ER_LMH << "    " << ER_inst << endl;
	File.close();
}

void plotER_PoreR()
{
	if(NowStep != AllStep) return;

	int headerflag = 0;

	char prefix[] = "./data";
	#ifdef WIN32
		if (_access(prefix , 0) == -1)	//如果文件夹不存在
			_mkdir(prefix);				//则创建
	#else
		if (access(prefix , 0) == -1)	//如果文件夹不存在
			mkdir(prefix, 0777);				//则创建
	#endif

	ofstream File("data/ER_PoreRadius.txt", ios::app);
	if(!File.good())
	{
		cout << "File ER_PoreRadius.txt open fail!" << endl;
		return;
	}

	if(!headerflag) File << "PoreRadius" << "	PoreRadius(um)" << "    ER" << "    ER_LMH(lm^-2h^-1)" << "    ER_inst(lm^-2h^-1)" << endl;
	
	File << PoreRadius << "	" << PoreRadius * DimLen * 10000 << "    " << ER << "    " << ER_LMH << "    " << ER_inst << endl;

	File.close();
}

__global__ void setTemperature(double * Te)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if (k > _PoreTop)
	{
		Te[I] = _Tcold;
	}
}

void plotInitField_origin()
{
	if(NowStep != 0) return;	

	ofstream file("data/InitField_Origin.txt");
	if(!file.good())
	{
		cout << "File InitField_Origin.txt open fail!" << endl;
		return;
	}

	//Header
	file << "X" << "\t" << "Y" << "\t" << "Density" << "\t" << "ChemicalPotential" << endl;

	//Solid
	for(int i = 0; i < DX; ++i)
	for(int k = 0; k < DZ; ++k)
	{
		if(Type[I(i, DY/2 ,k)] != FLUID)
			file << i << "\t" << k << "\t" << Pote[I(i,DY/2,k)] - 1<< "\t" << Pote[I(i,DY/2,k)] << endl;
		else //Fluid
		{
			file << i << "\t" << k << "\t" << Dens[I(i,DY/2 ,k)] << "\t" << Pote[I(i,DY/2 ,k)] << endl;
		}
	}
}

void SavePara()
{
	ofstream file("data/Para.txt");
	if(!file.good())
	{
		cout << "File Para.txt open fail!" << endl;
		return;
	}

	file << "Tau" << "	" << Tau << endl;
	file << "Tr" << "	" << Tr << endl;
	file << "DenG" << "	" << DenG << endl;
	file << "DenL" << "	" << DenL << endl;
	file << "Radius" << "	" << Radius << endl;
	file << "Interface Width" << "	" << Width << endl;
	file << "Ka" << "	" << Ka << endl;
	file << "K" << "	" << K << endl;
	// file << "Rey" << "	" << Rey << endl;
	file << "Viscosity" << "	" << Viscosity << endl;
	// file << "We" << "	" << We << endl;

	file << "AllStep" << "	" << AllStep << endl;
	file << "ShowStep" << "	" << ShowStep << endl;

	file << endl;

	file << "Geomeetry Parameters: " << endl;
	file << "DX" << "	" << DX << endl;
	file << "DY" << "	" << DY << endl;
	file << "DZ" << "	" << DZ << endl;
	file << "FilmThickness" << "	" << FilmThickness << endl;
	file << "Film from Z " << "	" << PoreBottom << " to " << PoreTop << endl;
	file << "PoreCenter(X, Y) " << "	" << PoreCenterX << " " << PoreCenterY << endl;
	file << "PoreRadius" << "	" << PoreRadius << endl;
	file << "HotSection Liquid from Z " << "	" << HotLiquidBottom << " to " << HotLiquidTop << endl;
	file << "ColdSection Liquid from Z " << "	" << ColdLiquidBottom << " to " << ColdLiquidTop << endl;

	file << endl;

	file << "ChemicalPotential Parameters: " << endl;
	file << "UpperPlate ChemicalPotential" << "	" << HydroPhilicPt << endl;
	file << "LowerPlate ChemicalPotential" << "	" << HydroPhobicPt << endl;
	file << "Film inner ChemicalPotential" << "	" << HydroPhilicPt << endl;
	file << "Pore ChemicalPotential" << "	" << HydroPhobicPt << endl;
	file << "Pore HyrdorPhobicPt Region Radius" << "	" << PtRadius << endl;

	file << endl;

	file << "Temperature Parameters: " << endl;
	file << "Hot Section Temperature(critical T / readl world T)" << "	" << Thot << "/" << Thot << endl;
	file << "Cold Section Temperature(critical T / readl world T)" << "	" << Tcold << "/" << Tcold << endl;
	// file << "T evolution time" << "	" << TevolveStep << "Step" << endl;
}

__global__ void SetFilmLowerSurfacePt(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if(Type[I] == FLUID) return;

	if(k >= _PoreBottom  && k <= _PoreTop)
	{
		Pote[I] =  Pote[I(DX/2, j,k)];
	}
}

__global__ void SetFilmLowerSurfacePt1(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k >= _PoreBottom  && k <= _PoreTop)
	{
		if(Type[I] != LEVEL1)  return ;
        double avg_pt= 0;
        double w = 0;
        for (int f = 1; f < DQ; ++f) 
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == FLUID)
            {
               avg_pt += Alpha[f] * Pote[pp];
               w += Alpha[f];
            }
        }
        Pote[I] = avg_pt/ w;
	}
}

__global__ void SetFilmLowerSurfacePt2(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k >= _PoreBottom  && k <= _PoreTop)
	{
		if(Type[I] != LEVEL2)  return ;

        double avg_pt= 0;
        double w = 0;

        for (int f = 1; f < DQ; ++f) 
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == LEVEL1)
            {
               avg_pt += Alpha[f] * Pote[pp];
               w += Alpha[f];
            }
        }
        Pote[I] = avg_pt/ w;
	}
}

__global__ void SetFilmLowerSurfacePt3(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k >= _PoreBottom  && k <= _PoreTop)
	{
		if(Type[I] != LEVEL3)  return ;

        double avg_pt= 0;
        double w = 0;

        for (int f = 1; f < DQ; ++f) 
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == LEVEL2)
            {
               avg_pt += Alpha[f] * Pote[pp];
               w += Alpha[f];
            }
        }
        Pote[I] = avg_pt/ w;
	}
}


__global__ void Plan2_SetFilmLowerSurfacePt1(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k == _PoreBottom && Type[I] != FLUID) 
		Pote[I] = Pote[I(i,j, k-1)];

	if ( k == _PoreTop && Type[I] != FLUID) 
		Pote[I] = Pote[I(i,j, k+1)];
}

__global__ void Plan2_SetFilmLowerSurfacePt2(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;

	if ( k == _PoreBottom + 1 && Type[I] != FLUID) 
		Pote[I] = Pote[I(i,j, k-1)];
	
	if ( k == _PoreTop - 1 && Type[I] != FLUID) 
		Pote[I] = Pote[I(i,j, k+1)];
}


__global__ void Plan2_SetFilmLowerSurfacePt3(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if(k < _PoreTop - 1 && k > _PoreBottom + 1) 
	{
		if(Type[I] != LEVEL1)  return;

        double avg_pt= 0;
        double w = 0;

        for (int f = 1; f < DQ; ++f) 
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == FLUID)
            {
               avg_pt += Alpha[f] * Pote[pp];
               w += Alpha[f];
            }
        }
        Pote[I] = avg_pt/ w;
	}
}

__global__ void Plan2_SetFilmLowerSurfacePt4(char * Type, double * Pote)
{
	GridIndex;  LineIndex; if(I >= DXYZ) return;
	if(k < _PoreTop - 1 && k > _PoreBottom + 1) 
	{
		if(Type[I] != LEVEL2)  return;


        for (int f = 1; f < 3; ++f)   //只考虑x方向
        {
            int xoffset = (i + Ex[f] + DX) % DX , yoffset = (j + Ey[f] + DY) % DY, zoffset = (k + Ez[f] + DZ) % DZ;
            const int pp =  xoffset * DY * DZ + yoffset * DZ + zoffset;

            if(Type [pp] == LEVEL1)
            {
				if(i < DX/2)
				{
					for(int idx_i = 0; idx_i <= i; ++idx_i)
					{
						Pote[I(idx_i, j, k)] = Pote[pp];
					}
				}
				else
				{
					for(int idx_i = i; idx_i < DX; ++idx_i)
					{
						Pote[I(idx_i, j, k)] = Pote[pp];
					}
				}
            }
        }
	}
}


void plotEosPressure(const string TestName)
{
	char prefix[] = "./data";
	#ifdef WIN32
		if (_access(prefix , 0) == -1)	//如果文件夹不存在
			_mkdir(prefix);				//则创建
	#else
		if (access(prefix , 0) == -1)	//如果文件夹不存在
			mkdir(prefix, 0777);				//则chmod
	#endif

	string FileName = "data/" + TestName + "_EosPressure_curve.txt";

	ofstream File(FileName.c_str() , ios::app);
	if(!File.good())
	{
		cout << "File EosPressure.txt open fail!" << endl;
		return;
	}

	File << "Y" << "    " << "Pressure" << "	" << "Pressure(Pa)" << endl;

	hipDeviceSynchronize();

	for(int k = 0; k < DZ; ++k)
	{
		if(Type[I(DX/2, DY/2, k)] != FLUID) continue;
		double Press = EosPressure(Dens[I(DX/2, DY/2, k)]);
		File << k << "    " << Press << "	" << Press * DimMass / DimLen / Sq(DimTime) /10  << endl;
	}

	File.close();
}